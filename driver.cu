#include "utils.cuh"
#include "runners.cuh"
#include <chrono>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <string>
#include <sstream>
#include <string_view>

using std::cout;
using std::endl;
using std::cin;

bool run_single_kernel(int kernel_choice, int rows, int cols, int inners, 
                      float* d_A, float* d_B, float* d_C, 
                      float* h_C, float* h_C_ref, int runs, 
                      bool verbose, bool check_results, std::string_view pattern) {
    bool res = false;

    // Reset d_C to zeros before each kernel
    cudaCheckError(hipMemset(d_C, 0, rows * cols * sizeof(float)));

    if (verbose) {
        cout << "Running kernel " << kernel_choice << ": " << get_kernel_name(kernel_choice);
        if (!check_results) {
            cout << " (Performance-only mode)";
        }
        cout << endl;
    }

    switch (kernel_choice){
    case 1: // Naive Implementation
        res = run_naive(rows, cols, inners, d_A, d_B, d_C, runs);
        break;
    case 2: // Global Memory Coalescing
        res = run_gmem_coalesce(rows, cols, inners, d_A, d_B, d_C, runs);
        break;
    case 3: // Shared Memory Blocks
        res = run_smem_blocking(rows, cols, inners, d_A, d_B, d_C, runs);
        break;
    case 4: // One Dimensional Blocktiling
        res = run_one_blocktiling(rows, cols, inners, d_A, d_B, d_C, runs);
        break;
    case 5: // Two Dimensional Blocktiling
        res = run_two_blocktiling(rows, cols, inners, d_A, d_B, d_C, runs);
        break;
    case 6: // Vectorized Memory Accessing
        if (check_results) {
            res = run_vectorized(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs);
        } else {
            res = run_vectorized_no_check(rows, cols, inners, d_A, d_B, d_C, runs);
        }
        break;
    case 7: // 1 Dimensional Vectorized Approach
        if (check_results) {
            res = run_1d_vec(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs);
        } else {
            res = run_1d_vec_no_check(rows, cols, inners, d_A, d_B, d_C, runs);
        }
        break;
    case 8: // Basic Warptiling
        if (check_results) {
            res = run_warptiling(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs);
        } else {
            res = run_warptiling_no_check(rows, cols, inners, d_A, d_B, d_C, runs);
        }
        break;
    case 9: // 1-Dimensional Warptiling
        if (check_results) {
            res = run_1d_warptiling(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs);
        } else {
            res = run_1d_warptiling_no_check(rows, cols, inners, d_A, d_B, d_C, runs);
        }
        break;
    case 10: // Emergent Sparsity Matrix Multiplication (our kernel)
        if (check_results) {
            res = run_esmm(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs);
        } else {
            res = run_esmm_no_check(rows, cols, inners, d_A, d_B, d_C, runs);
        }
        break;
    case 11: // Experimental warpskipping approach to ESMM
        if (check_results) {
            res = run_esmm_warpskipping(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs);
        } else {
            res = run_esmm_warpskipping_no_check(rows, cols, inners, d_A, d_B, d_C, runs);
        }
        break;
    case 12: // Experimental double buffered approach to ESMM
        if (check_results) {
            res = run_esmm_buffered(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs);
        } else {
            res = run_esmm_buffered_no_check(rows, cols, inners, d_A, d_B, d_C, runs);
        }
        break;
    case 13: // Experimental offset based A-Sparsity approach to ESMM
        if (check_results) {
            res = run_esmm_offsets(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs, pattern);
        } else {
            res = run_esmm_offsets_no_check(rows, cols, inners, d_A, d_B, d_C, runs, pattern);
        }
        break;
    case 14: // cuBlas
        if (check_results) {
            run_cuBlas(rows, cols, inners, d_A, d_B, d_C, h_C, runs);
        } else {
            run_cuBlas_no_check(rows, cols, inners, d_A, d_B, d_C, runs);
        }
        res = true; // Assume cuBLAS always succeeds
        break;
    default:
        cout << "Invalid kernel choice: " << kernel_choice << endl;
        return false;
    }

    if (verbose) {
        if (check_results) {
            cout << "  Status: " << (res ? "PASSED" : "FAILED") << endl;
        } else {
            cout << "  Status: COMPLETED (no verification)" << endl;
        }
    }

    return res;
}

int main(int argc, char *argv[]) {
    // Define Matrix Dims
    constexpr int rows = 4096;
    constexpr int cols = 4096;
    constexpr int inners = 4096;

    // Default values
    std::vector<int> kernel_choices = {10};
    int runs = 1;
    bool verbose = false;
    bool check_results = true;
    for (int i = 1; i < argc; i++) {
        std::string arg = argv[i];
        if (arg == "--help" || arg == "-h") {
            print_usage(argv[0]);
            return 0;
        } else if (arg == "--verbose" || arg == "-v") {
            verbose = true;
        } else if (arg == "--no-check" || arg == "-n") {
            check_results = false;
        } else if (i == 1) {
            if (isdigit(arg[0]) || arg == "all" || arg.find(',') != std::string::npos || arg.find('-') != std::string::npos) {
                kernel_choices = parse_kernel_selection(arg);
                if (kernel_choices.empty()) {
                    cout << "Error: Invalid kernel selection '" << arg << "'" << endl;
                    print_usage(argv[0]);
                    return 1;
                }
            }
        } else if (i == 2 && isdigit(arg[0])) {
            runs = atoi(argv[i]);
            if (runs <= 0) {
                cout << "Error: Number of runs must be positive" << endl;
                return 1;
            }
        }
    }

    if (verbose) {
        cout << "Matrix dimensions: " << rows << "x" << cols << " * " << cols << "x" << inners << endl;
        cout << "Number of runs per kernel: " << runs << endl;
        cout << "Result checking: " << (check_results ? "ENABLED" : "DISABLED") << endl;
        cout << "Kernels to run: ";
        for (size_t i = 0; i < kernel_choices.size(); i++) {
            cout << kernel_choices[i];
            if (i < kernel_choices.size() - 1) cout << ", ";
        }
        cout << endl << endl;
    }

    float *h_A = (float *)malloc(rows * inners * sizeof(float));
    float *h_B = (float *)malloc(inners * cols * sizeof(float));
    float *h_C = (float *)malloc(rows * cols * sizeof(float));
    float *h_C_ref = (float *)malloc(rows * cols * sizeof(float));

    constexpr std::string_view sparsity = "00000010";

    randomize_matrix_with_pattern(h_A, rows, inners, sparsity);
    randomize_matrix(h_B, inners, cols);
    memset(h_C, 0, rows * cols * sizeof(float));

    float *d_A, *d_B, *d_C;
    cudaCheckError(hipMalloc(&d_A, rows * inners * sizeof(float)));
    cudaCheckError(hipMalloc(&d_B, inners * cols * sizeof(float)));
    cudaCheckError(hipMalloc(&d_C, rows * cols * sizeof(float)));

    cudaCheckError(hipMemcpy(d_A, h_A, rows * inners * sizeof(float),
                              hipMemcpyHostToDevice));
    cudaCheckError(hipMemcpy(d_B, h_B, inners * cols * sizeof(float),
                              hipMemcpyHostToDevice));

    if (check_results) {
        if (verbose) cout << "Generating CPU reference solution..." << endl;
        matrixMultiplyCPU(h_A, h_B, h_C_ref, rows, cols, inners);
    } else {
        if (verbose) cout << "Skipping CPU reference solution (no-check mode)..." << endl;
    }

    int passed = 0;
    int total = kernel_choices.size();

    for (int kernel_choice : kernel_choices) {
        bool result = run_single_kernel(kernel_choice, rows, cols, inners,
                                       d_A, d_B, d_C, h_C, h_C_ref, runs, 
                                       verbose, check_results, sparsity);
        if (result || !check_results) passed++;

        if (!verbose) {
            cout << "Kernel " << kernel_choice << " (" << get_kernel_name(kernel_choice) << "): ";
            if (check_results) {
                cout << (result ? "PASSED" : "FAILED");
            } else {
                cout << "COMPLETED (no verification)";
            }
            cout << endl;
        }

        if (verbose && kernel_choice != kernel_choices.back()) {
            cout << endl;
        }
    }

    cout << endl << "Summary: " << passed << "/" << total << " kernels ";
    if (check_results) {
        cout << "passed" << endl;
    } else {
        cout << "completed (no verification)" << endl;
    }

    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_ref);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return (passed == total) ? 0 : 1;
}

