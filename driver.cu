#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <iomanip>
#include <hip/hip_runtime.h>
#include "utils.cuh"
#include "./kernels/1D_Blocktiling.cu"
#include "./kernels/multi.cu"
#include "./kernels/multi2.cu"
#include "./kernels/multi3.cu"
#include "./kernels/1D_Blocktiling.cu"
#include "./kernels/2D_Blocktiling.cu"
#include "./kernels/vectorized_blocktiling.cu"
#include <chrono>

#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line) {
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

#define SETUP \
	auto start = std::chrono::high_resolution_clock::now(); \
	auto end = std::chrono::high_resolution_clock::now(); \
	double total_time = 0.0f;
#define START start = std::chrono::high_resolution_clock::now();
#define END \
	end = std::chrono::high_resolution_clock::now(); \
	total_time += std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
#define RESULTS(kernel) \
	std::cout << "Average Speed of Kernel " << kernel << " (" << runs << " runs): "\
	<< std::fixed << std::setprecision(4) \
	<< (total_time / runs) / 1000.0f << " ms" << std::endl;
/* Simple function to iterate over a kernel and get data for a specific (or all
 * of them) */
void collect_data(int runs, int kernel, int rows, int cols, int inners, int blocksize, float* d_A, float* d_B, float* d_C) {
	// Integer corresponds to the version of multi
	SETUP
	switch (kernel) {
		case 1: {
			for (int i = 0; i < runs; i++) {
				START
				esmm_shmem_multi<<<dim3(CEIL_DIV(rows, blocksize), CEIL_DIV(cols, blocksize)), dim3(blocksize), blocksize * blocksize * 2 * sizeof(float)>>>(rows, cols, inners, blocksize, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
				hipMemset(d_C, 0, rows * cols * sizeof(float));
			}
			RESULTS("Multi")
			break;
		}
		case 2: {
			for (int i = 0; i < runs; i++) {
				START
				esmm_shmem_multi2<<<dim3(CEIL_DIV(rows, blocksize), CEIL_DIV(cols, blocksize)), dim3(blocksize), blocksize * blocksize * 2 * sizeof(float)>>>(rows, cols, inners, blocksize, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
				hipMemset(d_C, 0, rows * cols * sizeof(float));
				}
			RESULTS("Multi2")
			break;
		}
		case 3: {
			for (int i = 0; i < runs; i++) {
				START
				esmm_shmem_multi3<<<dim3(CEIL_DIV(rows, blocksize), CEIL_DIV(cols, blocksize)), dim3(blocksize), blocksize * blocksize * 2 * sizeof(float)>>>(rows, cols, inners, blocksize, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
				hipMemset(d_C, 0, rows * cols * sizeof(float));
			}
			RESULTS("Multi3")
			break;
		}
		case 4: {
						// 1D Blocktiling
			constexpr int BM = 128;
			constexpr int BN = 128;
			constexpr int BK = 8;
			constexpr int TM = 8;
			dim3 gridDim(CEIL_DIV(cols, BN), CEIL_DIV(rows, BM));
			dim3 blockDim(BN * BM / TM);
			for (int i = 0; i < runs; i++) {
				START
				one_blocktiling<BM, BN, BK, TM><<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
				hipMemset(d_C, 0, rows * cols * sizeof(float));
			}
			RESULTS("1D Blocktiling")
			break;
		}
		case 5: {
			// 2D Blocktiling
			constexpr int BM = 64;
			constexpr int BN = 64;
			constexpr int BK = 8;
			constexpr int TM = 8;
			constexpr int TN = 8;
			dim3 gridDim(CEIL_DIV(cols, BN), CEIL_DIV(rows, BM));
			dim3 blockDim(BM * BN / (TM * TN));
			for (int i = 0; i < runs; i++) {
				START
				two_blocktiling<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
				hipMemset(d_C, 0, rows * cols * sizeof(float));
			}
			RESULTS("2D Blocktiling")
			break;
		}
		case 6: {
			// Vectorized Blocktiling
			constexpr int BM = 64;
			constexpr int BN = 64;
			constexpr int BK = 8;
			constexpr int TM = 8;
			constexpr int TN = 8;
			dim3 gridDim(CEIL_DIV(cols, BN), CEIL_DIV(rows, BM));
			dim3 blockDim(BM * BN / (TM * TN));
			for (int i = 0; i < runs; i++) {
				START
				vectorized_blocktiling<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
				hipMemset(d_C, 0, rows * cols * sizeof(float));
			}
			RESULTS("Vectorized Blocktiling")
			break;
		}
		default:
			// Run all kernels
			collect_data(runs, 1, rows, cols, inners, blocksize, d_A, d_B, d_C);
			collect_data(runs, 2, rows, cols, inners, blocksize, d_A, d_B, d_C);
			collect_data(runs, 3, rows, cols, inners, blocksize, d_A, d_B, d_C);
			/* 1d blocktiling isnt working right now */ 
			//collect_data(runs, 4, rows, cols, inners, blocksize, d_A, d_B, d_C);
			//collect_data(runs, 5, rows, cols, inners, blocksize, d_A, d_B, d_C);
			collect_data(runs, 6, rows, cols, inners, blocksize, d_A, d_B, d_C);
	}
}

int main() {
		// Setup 
		constexpr int rows = 4096;
		constexpr int cols = 4096;
		constexpr int inners = 4096;
		constexpr int blocksize = 32;
		// Allocate host matrices
		float *h_A = (float*)malloc(rows * cols * sizeof(float));
		float *h_B = (float*)malloc(rows * cols * sizeof(float));
		float *h_C = (float*)malloc(rows * cols * sizeof(float));
		float *h_C_cpu = (float*)malloc(rows * cols * sizeof(float));

		// Generate random data
		randomize_matrix(h_A, rows, cols);
		randomize_matrix(h_B, rows, cols);

		// Allocate device matrices
		float *d_A, *d_B, *d_C;
		cudaCheckError(hipMalloc(&d_A, rows * cols * sizeof(float)));
		cudaCheckError(hipMalloc(&d_B, rows * cols * sizeof(float)));
		cudaCheckError(hipMalloc(&d_C, rows * cols * sizeof(float)));

		// Copy random data to device matrices
		cudaCheckError(hipMemcpy(d_A, h_A, rows * cols * sizeof(float), hipMemcpyHostToDevice));
		cudaCheckError(hipMemcpy(d_B, h_B, rows * cols * sizeof(float), hipMemcpyHostToDevice));

		collect_data(1, 0, rows, cols, inners, blocksize, d_A, d_B, d_C);

		// Verify GPU computation
		//bool correct = verifyResults(h_C, h_C_cpu, rows * cols);
		//printf("Matrix multiplication %s\n", correct ? "PASSED" : "FAILED");

		free(h_A);
		free(h_B);
		free(h_C);
		free(h_C_cpu);
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
		return 0;
}


