#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"
#include "./kernels/1D_Blocktiling.cu"

#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line) {
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

int main() {
	// Setup 
	constexpr int rows = 512;
	constexpr int cols = 512;
	constexpr int inners = 512;

	float *h_A = (float*)malloc(rows * cols * sizeof(float));
	float *h_B = (float*)malloc(rows * cols * sizeof(float));
	float *h_C = (float*)malloc(rows * cols * sizeof(float));
	float *h_C_cpu = (float*)malloc(rows * cols * sizeof(float));

	randomize_matrix(h_A, rows, cols);
	randomize_matrix(h_B, rows, cols);

	float *d_A, *d_B, *d_C;
	cudaCheckError(hipMalloc(&d_A, rows * cols * sizeof(float)));
	cudaCheckError(hipMalloc(&d_B, rows * cols * sizeof(float)));
	cudaCheckError(hipMalloc(&d_C, rows * cols * sizeof(float)));

	cudaCheckError(hipMemcpy(d_A, h_A, rows * cols * sizeof(float), hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpy(d_B, h_B, rows * cols * sizeof(float), hipMemcpyHostToDevice));

	hipMemset(d_C, 0, rows * cols * sizeof(float));

	//Run 1D blocktiling kernel
	constexpr int blockHeight = 64;
	constexpr int blockWidth = 64;
	constexpr int blockInner = 8;
	constexpr int resultsPerThread = 8;
	dim3 gridDim(CEIL_DIV(cols,blockWidth), CEIL_DIV(rows,blockHeight));
	dim3 blockDim((blockWidth * blockHeight) / resultsPerThread);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	sgemm1DBlocktiling<blockHeight,blockWidth,blockInner,resultsPerThread><<<gridDim, blockDim>>>(rows,cols,inners,d_A,d_B,d_C);
	hipEventRecord(stop);
	cudaCheckError(hipMemcpy(h_C, d_C, rows * cols * sizeof(float), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);


	//matMulBlockTiling<blockHeight, blockWidth, blockInner, resultsPerThread><<<gridDim, blockDim>>>(d_A, d_B, d_C, cols, inners);
	matrixMultiplyCPU(h_A, h_B, h_C_cpu, rows, cols);
	bool correct = verifyResults(h_C, h_C_cpu, rows * cols);
	printf("Matrix multiplication %s (%f ms)\n", correct ? "PASSED" : "FAILED", milliseconds);

	free(h_A);
	free(h_B);
	free(h_C);
	free(h_C_cpu);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	return 0;
}


