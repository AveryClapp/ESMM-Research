#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <iomanip>
#include <hip/hip_runtime.h>
#include "utils.cuh"
#include "./kernels/1D_Blocktiling.cu"
#include "./kernels/multi.cu"
#include "./kernels/multi2.cu"
#include "./kernels/multi3.cu"
#include <chrono>

#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line) {
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

#define SETUP \
	auto start = std::chrono::high_resolution_clock::now(); \
	auto end = std::chrono::high_resolution_clock::now(); \
	double total_time = 0.0f;
#define START start = std::chrono::high_resolution_clock::now();
#define END \
	end = std::chrono::high_resolution_clock::now(); \
	total_time += std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
#define RESULTS(kernel) \
	std::cout << "Average Speed of Kernel " << kernel << " (" << runs << " runs): "\
	<< std::fixed << std::setprecision(4) \
	<< (total_time / runs) / 1000.0f << " ms" << std::endl;
/* Simple function to iterate over a kernel and get data for a specific (or all
 * of them) */
void collect_data(int runs, int kernel, int rows, int cols, int inners, int blocksize, float* d_A, float* d_B, float* d_C) {
	// Integer corresponds to the version of multi
	SETUP
	switch (kernel) {
		case 1: {
			for (int i = 0; i < runs; i++) {
				START
				esmm_shmem_multi<<<dim3(CEIL_DIV(rows, blocksize), CEIL_DIV(cols, blocksize)), dim3(blocksize), blocksize * blocksize * 2 * sizeof(float)>>>(rows, cols, inners, blocksize, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
				hipMemset(d_C, 0, rows * cols * sizeof(float));
			}
			RESULTS("Multi")
			break;
		}
		case 2: {
			for (int i = 0; i < runs; i++) {
				START
				esmm_shmem_multi2<<<dim3(CEIL_DIV(rows, blocksize), CEIL_DIV(cols, blocksize)), dim3(blocksize), blocksize * blocksize * 2 * sizeof(float)>>>(rows, cols, inners, blocksize, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
				hipMemset(d_C, 0, rows * cols * sizeof(float));
				}
			RESULTS("Multi2")
			break;
		}
		case 3: {
			for (int i = 0; i < runs; i++) {
				START
				esmm_shmem_multi3<<<dim3(CEIL_DIV(rows, blocksize), CEIL_DIV(cols, blocksize)), dim3(blocksize), blocksize * blocksize * 2 * sizeof(float)>>>(rows, cols, inners, blocksize, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
				hipMemset(d_C, 0, rows * cols * sizeof(float));
			}
			RESULTS("Multi3")
			break;
		}
		default:
			// Run all kernels
			collect_data(runs, 1, rows, cols, inners, blocksize, d_A, d_B, d_C);
			collect_data(runs, 2, rows, cols, inners, blocksize, d_A, d_B, d_C);
			collect_data(runs, 3, rows, cols, inners, blocksize, d_A, d_B, d_C);
		}
}

int main() {
		// Setup 
		constexpr int rows = 4096;
		constexpr int cols = 4096;
		constexpr int inners = 4096;
		constexpr int blocksize = 32;
		// Allocate host matrices
		float *h_A = (float*)malloc(rows * cols * sizeof(float));
		float *h_B = (float*)malloc(rows * cols * sizeof(float));
		float *h_C = (float*)malloc(rows * cols * sizeof(float));
		float *h_C_cpu = (float*)malloc(rows * cols * sizeof(float));

		// Generate random data
		randomize_matrix(h_A, rows, cols);
		randomize_matrix(h_B, rows, cols);

		// Allocate device matrices
		float *d_A, *d_B, *d_C;
		cudaCheckError(hipMalloc(&d_A, rows * cols * sizeof(float)));
		cudaCheckError(hipMalloc(&d_B, rows * cols * sizeof(float)));
		cudaCheckError(hipMalloc(&d_C, rows * cols * sizeof(float)));

		// Copy random data to device matrices
		cudaCheckError(hipMemcpy(d_A, h_A, rows * cols * sizeof(float), hipMemcpyHostToDevice));
		cudaCheckError(hipMemcpy(d_B, h_B, rows * cols * sizeof(float), hipMemcpyHostToDevice));
	
		collect_data(10, 4, rows, cols, inners, blocksize, d_A, d_B, d_C);

		// Verify GPU computation
		//bool correct = verifyResults(h_C, h_C_cpu, rows * cols);
		//printf("Matrix multiplication %s\n", correct ? "PASSED" : "FAILED");

		free(h_A);
		free(h_B);
		free(h_C);
		free(h_C_cpu);
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
		return 0;
}


