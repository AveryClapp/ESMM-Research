#include "./kernels/1D_Blocktiling.cu"
#include "./kernels/2D_Blocktiling.cu"
#include "./kernels/basic.cu"
#include "./kernels/gmem_coalesce.cu"
#include "./kernels/smem_blocking.cu"
#include "./kernels/vectorized_blocktiling.cu"
#include "./kernels/warptiling.cu"
#include "./kernels/1D_vec.cu"
#include "utils.cuh"
#include <chrono>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

#define cudaCheckError(ans)                                                    \
  {                                                                            \
    cudaAssert((ans), __FILE__, __LINE__);                                     \
  }
inline void cudaAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file,
            line);
    exit(code);
  }
}

#define SETUP                                                                  \
  auto start = std::chrono::high_resolution_clock::now();                      \
  auto end = std::chrono::high_resolution_clock::now();                        \
  double total_time = 0.0f;
#define START start = std::chrono::high_resolution_clock::now();
#define END                                                                    \
  end = std::chrono::high_resolution_clock::now();                             \
  total_time +=                                                                \
      std::chrono::duration_cast<std::chrono::microseconds>(end - start)       \
          .count();
#define RESULTS(kernel)                                                        \
  std::cout << "Average Speed of Kernel " << kernel << " (" << runs            \
            << " runs): " << std::fixed << std::setprecision(4)                \
            << (total_time / runs) / 1000.0f << " ms" << std::endl;

const uint K10_NUM_THREADS = 128;
const uint K10_BN = 128;
const uint K10_BM = 128;
const uint K10_BK = 16;
const uint K10_WN = 64;
const uint K10_WM = 64;
const uint K10_WNITER = 4;
const uint K10_TN = 4;
const uint K10_TM = 8;


void run_naive(int rows, int cols, int inners, float *d_A, float *d_B,
               float *d_C, int runs) {
  dim3 gridDim(CEIL_DIV(cols, 32), CEIL_DIV(rows, 32));
  dim3 blockDim(32, 32);
  SETUP
  for (int i = 0; i < runs; i++) {
    START
    basic<<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
    END hipDeviceSynchronize();
  }
  RESULTS("Naive");
}

void run_gmem_coalesce(int rows, int cols, int inners, float *d_A, float *d_B,
                       float *d_C, int runs) {
  dim3 gridDim(CEIL_DIV(cols, 32), CEIL_DIV(rows, 32));
  dim3 blockDim(32, 32);
  SETUP
  for (int i = 0; i < runs; i++) {
    START
    gmem_coalesce<32><<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
    END hipDeviceSynchronize();
  }
  RESULTS("GMEM Coalescing");
}

void run_smem_blocking(int rows, int cols, int inners, float *d_A, float *d_B,
                       float *d_C, int runs) {
  dim3 gridDim(CEIL_DIV(cols, 32), CEIL_DIV(rows, 32));
  dim3 blockDim(32, 32);
  SETUP
  for (int i = 0; i < runs; i++) {
    START
    smem_blocking<32><<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
    END hipDeviceSynchronize();
  }
  RESULTS("SMEM Blocking");
}

void run_one_blocktiling(int rows, int cols, int inners, float *d_A, float *d_B,
                         float *d_C, int runs) {
  constexpr int BM = 64;
  constexpr int BN = 64;
  constexpr int BK = 8;
  constexpr int TM = 8;
  dim3 gridDim(CEIL_DIV(cols, BN), CEIL_DIV(rows, BM));
  dim3 blockDim(BN * BM / TM);
  SETUP
  for (int i = 0; i < runs; i++) {
    START
    one_blocktiling<BM, BN, BK, TM>
        <<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
    END hipDeviceSynchronize();
  }
  RESULTS("1D Blocktiling")
}

bool run_two_blocktiling(int rows, int cols, int inners, float *d_A, float *d_B,
                         float *d_C, float *h_C, float*  h_C_ref, int runs) {

  constexpr int BM = 64;
  constexpr int BN = 64;
  constexpr int BK = 8;
  constexpr int TM = 8;
  constexpr int TN = 1;
  dim3 gridDim(CEIL_DIV(cols, BN), CEIL_DIV(rows, BM));
  dim3 blockDim(BM * BN / (TM * TN));
  SETUP
  for (int i = 0; i < runs; i++) {
    START
    two_blocktiling<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
    END hipDeviceSynchronize();
  }
  RESULTS("2D Blocktiling")
  hipMemcpy(h_C, d_C, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
	printf("CUDA error: %s\n", hipGetErrorString(error));
  }
 return verifyResults(h_C, h_C_ref, rows * cols);
}

bool run_1d_vec(int rows, int cols, int inners, float *d_A, float *d_B,
                         float *d_C, float *h_C, float *h_C_ref, int runs) {
  constexpr int BM = 64;
  constexpr int BN = 64;
  constexpr int BK = 8;
  constexpr int TM = 8;
  constexpr int TN = 1;
  dim3 gridDim(CEIL_DIV(cols, BN), CEIL_DIV(rows, BM));
  dim3 blockDim(BM * BN / (TM * TN));
  SETUP
  for (int i = 0; i < runs; i++) {
    START
    one_d_vec<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
    END 
	hipDeviceSynchronize();
  }
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
	printf("CUDA error: %s\n", hipGetErrorString(error));
  }
  RESULTS("1D Vectorized Blocktiling")
  hipMemcpy(h_C, d_C, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
  return verifyResults(h_C, h_C_ref, rows * cols);
}

bool run_vectorized(int rows, int cols, int inners, float *d_A, float *d_B,
                    float *d_C, float *h_C, float *h_C_ref, int runs) {
  constexpr int BM = 64;
  constexpr int BN = 64;
  constexpr int BK = 8;
  constexpr int TM = 8;
  constexpr int TN = 1;
  dim3 gridDim(CEIL_DIV(cols, BN), CEIL_DIV(rows, BM));
  dim3 blockDim(BM * BN / (TM * TN));
  SETUP
  for (int i = 0; i < runs; i++) {
    START
    vectorized_blocktiling<BM, BN, BK, TM, TN>
        <<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
    END
	hipDeviceSynchronize();
  }
  RESULTS("2D Vectorized Blocktiling")
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
	printf("CUDA error: %s\n", hipGetErrorString(error));
  }
  hipMemcpy(h_C, d_C, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
  return verifyResults(h_C, h_C_ref, rows * cols);

}

bool run_warptiling(int rows, int cols, int inners, float *d_A, float *d_B,
                    float *d_C, float *h_C, float *h_C_ref, int runs) {
  // Setup cuda timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  dim3 blockDim(K10_NUM_THREADS);
  dim3 gridDim(CEIL_DIV(cols, K10_BN), CEIL_DIV(rows, K10_BM));

  // Initialize C to zeros
  hipMemset(d_C, 0, rows * cols * sizeof(float));

  for (int i = 0; i < runs; i++) {
    hipEventRecord(start);
    warptiling<K10_BM, K10_BN, K10_BK, K10_WM, K10_WN, K10_WNITER, K10_TM, K10_TN, K10_NUM_THREADS><<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
	hipEventRecord(stop);
    hipDeviceSynchronize();
	hipEventSynchronize(stop);
  }
  
  hipMemcpy(h_C, d_C, rows * cols * sizeof(float), hipMemcpyDeviceToHost);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
	//return false;
	std::cout << "FAIL" << std::endl;
  } else {
	//bool success = verifyResults(h_C, h_C_ref, rows * cols);
	//if (!success) {
		//return false;
	//}
	float time = 0;
	hipEventElapsedTime(&time, start, stop);
	std::cout << time << " ms" << std::endl;

	//return true;
  }
  hipEventDestroy(start);
  hipEventDestroy(stop);
}

void run_cuBlas(int rows, int cols, int inners, float *d_A, float *d_B,
                float *d_C, float *h_C, int runs) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float alpha = 1.0f;
  float beta = 0.0f;

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cols, rows, inners, &alpha, d_B,
              cols, d_A, inners, &beta, d_C, cols);

  SETUP
  for (int i = 0; i < runs; i++) {
    START
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cols, rows, inners, &alpha,
                d_B, cols, d_A, inners, &beta, d_C, cols);
    END hipDeviceSynchronize();
  }
  cudaCheckError(hipMemcpy(h_C, d_C, rows * cols * sizeof(float),
                            hipMemcpyDeviceToHost));
  RESULTS("cuBLAS")
  hipblasDestroy(handle);
}

int main(int argc, char *argv[]) {
  // Setup
  constexpr int rows = 1024;
  constexpr int cols = 1024;
  constexpr int inners = 1024;
  int kernel_choice = 10; // Default to warptiling
  int runs = 1;          // Default number of runs

  // Parse command line arguments
  if (argc > 1) {
    kernel_choice = atoi(argv[1]);
  }

  if (argc > 2) {
    runs = atoi(argv[2]);
  }

  // Allocate host matrices
  float *h_A = (float *)malloc(rows * inners * sizeof(float));
  float *h_B = (float *)malloc(inners * cols * sizeof(float));
  float *h_C = (float *)malloc(rows * cols * sizeof(float));
  float *h_C_ref = (float *)malloc(rows * cols * sizeof(float));

  // Generate random data
  randomize_matrix(h_A, rows, inners);
  randomize_matrix(h_B, inners, cols);

  // Set h_C to zeros
  memset(h_C, 0, rows * cols * sizeof(float));

  // Allocate device matrices
  float *d_A, *d_B, *d_C;
  cudaCheckError(hipMalloc(&d_A, rows * inners * sizeof(float)));
  cudaCheckError(hipMalloc(&d_B, inners * cols * sizeof(float)));
  cudaCheckError(hipMalloc(&d_C, rows * cols * sizeof(float)));

  // Copy random data to device matrices
  cudaCheckError(hipMemcpy(d_A, h_A, rows * inners * sizeof(float),
                            hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(d_B, h_B, inners * cols * sizeof(float),
                            hipMemcpyHostToDevice));

  // Generate reference solution on CPU
  //matrixMultiplyCPU(h_A, h_B, h_C_ref, rows, cols, inners);

  // Initialize d_C to zeros
  cudaCheckError(hipMemset(d_C, 0, rows * cols * sizeof(float)));

  bool verificationResult = true;

  // Choose kernel based on input
  switch (kernel_choice) {
  case 1:
    run_naive(rows, cols, inners, d_A, d_B, d_C, runs);
    break;
  case 2:
    run_gmem_coalesce(rows, cols, inners, d_A, d_B, d_C, runs);
    break;
  case 3:
    run_smem_blocking(rows, cols, inners, d_A, d_B, d_C, runs);
    break;
  case 4:
    //run_one_blocktiling(rows, cols, inners, d_A, d_B, d_C, runs);
	std::cout << run_two_blocktiling(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs) << std::endl;
    break;
  case 5:
	std::cout << run_1d_vec(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs) << std::endl;
    break;
  case 6:
	std::cout << run_vectorized(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs) << std::endl;
    break;
  case 10:
	run_warptiling(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs);
    break;
  case 11:
    run_cuBlas(rows, cols, inners, d_A, d_B, d_C, h_C, runs);
    break;
  case 12:
    run_naive(rows, cols, inners, d_A, d_B, d_C, runs);
    run_gmem_coalesce(rows, cols, inners, d_A, d_B, d_C, runs);
    run_smem_blocking(rows, cols, inners, d_A, d_B, d_C, runs);
    run_one_blocktiling(rows, cols, inners, d_A, d_B, d_C, runs);
    run_two_blocktiling(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs);
	run_1d_vec(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs);
    //run_vectorized(rows, cols, inners, d_A, d_B, d_C, runs);
    //run_warptiling(rows, cols, inners, d_A, d_B, d_C, h_C, h_C_ref, runs);
    break;
  default:
    std::cout << "Invalid kernel choice. Using warptiling (10) by default."
              << std::endl;
    break;
  }

  // Clean up
  free(h_A);
  free(h_B);
  free(h_C);
  free(h_C_ref);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  return verificationResult ? 0 : 1;
}
