#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"
#include "kernels.cu"

#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line) {
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}


void matrixMultiplyCPU(float* A, float* B, float* C) {               
	for (int row = 0; row < A_ROWS; row++) { 
		for (int col = 0; col < B_COLS; col++) {
			float sum = 0.0f;
			for (int i = 0; i < A_COLS; i++) {
				sum += A[row * A_COLS + i] * B[i * B_COLS + col];
			}
		C[row * C_COLS + col] = sum;
		}
	}
}

// Verify results
bool verifyResults(float* gpuResult, float* cpuResult, int size, float tolerance = 1e-3) {
	for (int i = 0; i < size; i++) {
		if (fabs(gpuResult[i] - cpuResult[i]) > tolerance) {
			printf("Mismatch at position %d: GPU = %f, CPU = %f\n",
				i, gpuResult[i], cpuResult[i]);
			return false;
		}
	}
	return true;
}

int main() {
	float *h_A = (float*)malloc(A_ROWS * A_COLS * sizeof(float));
	float *h_B = (float*)malloc(B_ROWS * B_COLS * sizeof(float));
	float *h_C = (float*)malloc(C_ROWS * C_COLS * sizeof(float));
	float *h_C_cpu = (float*)malloc(C_ROWS * C_COLS * sizeof(float));

	randomize_matrix(h_A, A_ROWS, A_COLS);
	randomize_matrix(h_B, B_ROWS, B_COLS);

	float *d_A, *d_B, *d_C;
	cudaCheckError(hipMalloc(&d_A, A_ROWS * A_COLS * sizeof(float)));
	cudaCheckError(hipMalloc(&d_B, B_ROWS * B_COLS * sizeof(float)));
	cudaCheckError(hipMalloc(&d_C, C_ROWS * C_COLS * sizeof(float)));

	cudaCheckError(hipMemcpy(d_A, h_A, A_ROWS * A_COLS * sizeof(float), hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpy(d_B, h_B, B_ROWS * B_COLS * sizeof(float), hipMemcpyHostToDevice));

	hipFree(0);
	hipMemset(d_C, 0, C_ROWS * C_COLS * sizeof(float));
	// 1024 blocks of 32
	dim3 gridDim(1024);
	dim3 blockDim(32);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	matrixMultiplyKernelSHMEM<<<gridDim, blockDim>>>(d_A, d_B, d_C);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	cudaCheckError(hipMemcpy(h_C, d_C, C_ROWS * C_COLS * sizeof(float), hipMemcpyDeviceToHost));

	float time = 0.0f;
	hipEventElapsedTime(&time, start, stop);

	std::cout << "GPU Timing: " << time << " ms" << std::endl;
	matrixMultiplyCPU(h_A, h_B, h_C_cpu);

	bool correct = verifyResults(h_C, h_C_cpu, C_ROWS * C_COLS);
		//printf("Matrix
		//multiplication
		//%s\n",
		//correct ?
		//"PASSED" :
		//"FAILED");

	free(h_A);
	free(h_B);
	free(h_C);
	free(h_C_cpu);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}


