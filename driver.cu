#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"
#include "./kernels/1D_Blocktiling.cu"

#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line) {
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

int main() {
	// Setup 
	constexpr int rows = 128;
	constexpr int cols = 128;
	constexpr int inners = 128;

	float *h_A = (float*)malloc(rows * cols * sizeof(float));
	float *h_B = (float*)malloc(rows * cols * sizeof(float));
	float *h_C = (float*)malloc(rows * cols * sizeof(float));
	float *h_C_cpu = (float*)malloc(rows * cols * sizeof(float));

	randomize_matrix(h_A, rows, cols);
	randomize_matrix(h_B, rows, cols);

	float *d_A, *d_B, *d_C;
	cudaCheckError(hipMalloc(&d_A, rows * cols * sizeof(float)));
	cudaCheckError(hipMalloc(&d_B, rows * cols * sizeof(float)));
	cudaCheckError(hipMalloc(&d_C, rows * cols * sizeof(float)));

	cudaCheckError(hipMemcpy(d_A, h_A, rows * cols * sizeof(float), hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpy(d_B, h_B, rows * cols * sizeof(float), hipMemcpyHostToDevice));

	hipMemset(d_C, 0, rows * cols * sizeof(float));

	//Run 1D blocktiling kernel
	constexpr int blockHeight = 32; // Height of tiled block	
	constexpr int blockWidth = 32; // Width of tiled block
	constexpr int blockInner = 8; // width of tiled block
	constexpr int resultsPerThread = 8;
	dim3 gridDim(CEIL_DIV(cols,blockWidth),CEIL_DIV(rows,blockHeight));
	dim3 blockDim((blockHeight * blockWidth)/blockInner);
	matMulBlocktiling<blockHeight, blockWidth, blockInner, resultsPerThread><<<gridDim, blockDim>>>(d_A, d_B, d_C, cols, inners);
	cudaCheckError(hipMemcpy(h_C, d_C, rows * cols * sizeof(float), hipMemcpyDeviceToHost));
	bool correct = verifyResults(h_C, h_C_cpu, rows * cols);
	printf("Matrix multiplication %s\n", correct ? "PASSED" : "FAILED");

	free(h_A);
	free(h_B);
	free(h_C);
	free(h_C_cpu);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}


