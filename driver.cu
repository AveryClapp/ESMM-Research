#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"
#include "./kernels/1D_Blocktiling.cu"
#include "./kernels/multi.cu"
#include "./kernels/multi2.cu"
#include "./kernels/multi3.cu"
#include <chrono>

#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line) {
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

#define START auto  start = std::chrono::high_resolution_clock::now();
#define END(kernel) \
	auto end = std::chrono::high_resolution_clock::now(); \
	auto time = std::chrono::duration_cast<std::chrono::microseconds>(end - start); \
	std::cout << #kernel << " took " << time.count() / 1000.0 << " milliseconds" << std::endl;

int main() {
	// Setup 
	constexpr int rows = 1024;
	constexpr int cols = 1024;
	constexpr int inners = 1024;
	constexpr int blocksize = 32;
	// Allocate host matrices
	float *h_A = (float*)malloc(rows * cols * sizeof(float));
	float *h_B = (float*)malloc(rows * cols * sizeof(float));
	float *h_C = (float*)malloc(rows * cols * sizeof(float));
	float *h_C_cpu = (float*)malloc(rows * cols * sizeof(float));

	// Generate random data
	randomize_matrix(h_A, rows, cols);
	randomize_matrix(h_B, rows, cols);

	// Allocate device matrices
	float *d_A, *d_B, *d_C;
	cudaCheckError(hipMalloc(&d_A, rows * cols * sizeof(float)));
	cudaCheckError(hipMalloc(&d_B, rows * cols * sizeof(float)));
	cudaCheckError(hipMalloc(&d_C, rows * cols * sizeof(float)));

	// Copy random data to device matrices
	cudaCheckError(hipMemcpy(d_A, h_A, rows * cols * sizeof(float), hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpy(d_B, h_B, rows * cols * sizeof(float), hipMemcpyHostToDevice));

	// Reset output matrix to 0
	hipMemset(d_C, 0, rows * cols * sizeof(float));

	//Launch kernel and record elapsed time
	START;		
	esmm_shmem_multi3<<<dim3(CEIL_DIV(rows, blocksize), CEIL_DIV(cols, blocksize)), dim3(blocksize), blocksize * blocksize * 2 * sizeof(float)>>>(rows, cols, inners, blocksize, d_A, d_B, d_C);
	END("Multi")
	
	// Cleanup and copy GPU data
	cudaCheckError(hipGetLastError());
	cudaCheckError(hipDeviceSynchronize());
	cudaCheckError(hipMemcpy(h_C, d_C, rows * cols * sizeof(float), hipMemcpyDeviceToHost));
	hipMemset(d_C, 0, rows * cols * sizeof(float));

	// Verify GPU computation
	matrixMultiplyCPU(h_A, h_B, h_C_cpu, rows, cols);
	bool correct = verifyResults(h_C, h_C_cpu, rows * cols);
	printf("Matrix multiplication %s\n", correct ? "PASSED" : "FAILED");

	free(h_A);
	free(h_B);
	free(h_C);
	free(h_C_cpu);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}


