#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "utils.cuh"
#include "./kernels/basic.cu"
#include "./kernels/gmem_coalesce.cu"
#include "./kernels/smem_blocking.cu"
#include "./kernels/1D_Blocktiling.cu"
#include "./kernels/1D_Blocktiling.cu"
#include "./kernels/2D_Blocktiling.cu"
#include "./kernels/vectorized_blocktiling.cu"
#include "./kernels/warptiling.cu"
#include <chrono>

#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line) {
	if (code != hipSuccess) {
		fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

#define SETUP \
	auto start = std::chrono::high_resolution_clock::now(); \
	auto end = std::chrono::high_resolution_clock::now(); \
	double total_time = 0.0f;
#define START start = std::chrono::high_resolution_clock::now();
#define END \
	end = std::chrono::high_resolution_clock::now(); \
	total_time += std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
#define RESULTS(kernel) \
	std::cout << "Average Speed of Kernel " << kernel << " (" << runs << " runs): "\
	<< std::fixed << std::setprecision(4) \
	<< (total_time / runs) / 1000.0f << " ms" << std::endl;

/* Simple function to iterate over a kernel and get data for a specific (or all
 * of them) */
void collect_data(int runs, int kernel, int rows, int cols, int inners, int blocksize, float* d_A, float* d_B, float* d_C, float* h_C, float* h_C_cpu) {
	// Integer corresponds to the version of multi
	SETUP
	switch (kernel) {
		case 1: {
			dim3 gridDim(CEIL_DIV(cols, 32), CEIL_DIV(rows, 32));
			dim3 blockDim(32, 32);
			for (int i = 0; i < runs; i++) {
				START
				basic<<<gridDim, blockDim>>>(rows,cols,inners,d_A,d_B,d_C);
				END
				hipDeviceSynchronize();
			}
			RESULTS("Naive");
			break;
		}
		case 2: {
			dim3 gridDim(CEIL_DIV(cols, 32), CEIL_DIV(rows, 32));
			dim3 blockDim(32, 32);
			for (int i = 0; i < runs; i++) {
				START
				gmem_coalesce<32><<<gridDim, blockDim>>>(rows,cols,inners,d_A,d_B,d_C);
				END
				hipDeviceSynchronize();
			}
			RESULTS("GMEM Coalescing");		
			break;
		}
		case 3: {
			dim3 gridDim(CEIL_DIV(cols, 32), CEIL_DIV(rows, 32));
			dim3 blockDim(32, 32);
			for (int i = 0; i < runs; i++) {
				START
				smem_blocking<32><<<gridDim, blockDim>>>(rows,cols,inners,d_A,d_B,d_C);
				END
				hipDeviceSynchronize();
			}
			RESULTS("SMEM Blocking");		
			break;
		}
		case 4: {
			// 1D Blocktiling
			constexpr int BM = 64;
			constexpr int BN = 64;
			constexpr int BK = 8;
			constexpr int TM = 8;
			dim3 gridDim(CEIL_DIV(cols, BN), CEIL_DIV(rows, BM));
			dim3 blockDim(BN * BM / TM);
			for (int i = 0; i < runs; i++) {
				START
				one_blocktiling<BM, BN, BK, TM><<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
			}
			RESULTS("1D Blocktiling")
			break;
		}
		case 5: {
			// 2D Blocktiling
			constexpr int BM = 128;
			constexpr int BN = 128;
			constexpr int BK = 8;
			constexpr int TM = 8;
			constexpr int TN = 8;
			dim3 gridDim(CEIL_DIV(cols, BN), CEIL_DIV(rows, BM));
			dim3 blockDim(BM * BN / (TM * TN));
			for (int i = 0; i < runs; i++) {
				START
				two_blocktiling<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
			}
			RESULTS("2D Blocktiling")
			break;
		}
		case 6: {
			// Vectorized Blocktiling
			constexpr int BM = 128;
			constexpr int BN = 128;
			constexpr int BK = 8;
			constexpr int TM = 8;
			constexpr int TN = 8;
			dim3 gridDim(CEIL_DIV(cols, BN), CEIL_DIV(rows, BM));
			dim3 blockDim(BM * BN / (TM * TN));
			for (int i = 0; i < runs; i++) {
				START
				vectorized_blocktiling<BM, BN, BK, TM, TN><<<gridDim, blockDim>>>(rows, cols, inners, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
			}
			RESULTS("Vectorized Blocktiling")
			break;
		}
		case 7: {
			const uint K10_NUM_THREADS = 128;
			const uint K10_BN = 128;
			const uint K10_BM = 128;
			const uint K10_BK = 16;
			const uint K10_WN = 64;
			const uint K10_WM = 64;
			const uint K10_WNITER = 4;
			const uint K10_TN = 4;
			const uint K10_TM = 8;

  			dim3 blockDim(K10_NUM_THREADS);
    		constexpr uint NUM_WARPS = K10_NUM_THREADS / 32;

			constexpr uint K10_WMITER = (K10_WM * K10_WN) / (32 * K10_TM * K10_TN * K10_WNITER);
			
			dim3 gridDim(CEIL_DIV(cols, K10_BN), CEIL_DIV(rows, K10_BM));
			for (int i = 0; i < runs; i++) {
				START
				warptiling<K10_BM, K10_BN, K10_BK, K10_WM, K10_WN, K10_WNITER, K10_TM, K10_TN, K10_NUM_THREADS> <<<gridDim, blockDim>>>(cols, rows, inners, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
			}
			RESULTS("Warptiling")
			break;

		}
		case 8: {
			hipblasHandle_t handle;
    		hipblasCreate(&handle);
	   		float alpha = 1.0f;
		    float beta = 0.0f;

			hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cols, rows, inners, &alpha, d_B, cols, d_A, inners, &beta, d_C, cols);
				
			for (int i = 0; i < runs; i++) {
				START
				hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, cols, rows, inners, &alpha, d_B, cols, d_A, inners, &beta, d_C, cols);
				END
				hipDeviceSynchronize();
				cudaCheckError(hipMemcpy(h_C, d_C, rows * cols * sizeof(float), hipMemcpyDeviceToHost));
				hipMemset(d_C, 0, rows * cols * sizeof(float));
			}
			RESULTS("cuBLAS")	
			hipblasDestroy(handle);	
			break;
		}
		default:
			// Run all kernels
			collect_data(runs, 1, rows, cols, inners, blocksize, d_A, d_B, d_C, h_C, h_C_cpu);
			collect_data(runs, 2, rows, cols, inners, blocksize, d_A, d_B, d_C, h_C, h_C_cpu);
			collect_data(runs, 3, rows, cols, inners, blocksize, d_A, d_B, d_C, h_C, h_C_cpu);
			collect_data(runs, 4, rows, cols, inners, blocksize, d_A, d_B, d_C, h_C, h_C_cpu);
			collect_data(runs, 5, rows, cols, inners, blocksize, d_A, d_B, d_C, h_C, h_C_cpu);
			collect_data(runs, 6, rows, cols, inners, blocksize, d_A, d_B, d_C, h_C, h_C_cpu);
			collect_data(runs, 7, rows, cols, inners, blocksize, d_A, d_B, d_C, h_C, h_C_cpu);
			collect_data(runs, 8, rows, cols, inners, blocksize, d_A, d_B, d_C, h_C, h_C_cpu);
	}
}

int main() {
		// Setup 
		constexpr int rows = 1024;
		constexpr int cols = 1024;
		constexpr int inners = 1024;
		constexpr int blocksize = 32;
		// Allocate host matrices
		float *h_A = (float*)malloc(rows * cols * sizeof(float));
		float *h_B = (float*)malloc(rows * cols * sizeof(float));
		float *h_C = (float*)malloc(rows * cols * sizeof(float));
		float *h_C_cpu = (float*)malloc(rows * cols * sizeof(float));

		// Generate random data
		randomize_matrix(h_A, rows, cols);
		randomize_matrix(h_B, rows, cols);

		// Allocate device matrices
		float *d_A, *d_B, *d_C;
		cudaCheckError(hipMalloc(&d_A, rows * cols * sizeof(float)));
		cudaCheckError(hipMalloc(&d_B, rows * cols * sizeof(float)));
		cudaCheckError(hipMalloc(&d_C, rows * cols * sizeof(float)));

		// Copy random data to device matrices
		cudaCheckError(hipMemcpy(d_A, h_A, rows * cols * sizeof(float), hipMemcpyHostToDevice));
		cudaCheckError(hipMemcpy(d_B, h_B, rows * cols * sizeof(float), hipMemcpyHostToDevice));

		matrixMultiplyCPU(h_A, h_B, h_C_cpu, rows, cols);
		// Run kernels
		/* 0 - all
		 * 1 - Naive
		 * 2 - GMEM Coalescing
		 * 3 - SMEM Blocking
		 * 4 - 1d
		 * 5 - 2d
		 * 6 - vectorized
		 * 7 - warptiling
		 * 8 - cuBLAS
		*/
		collect_data(1, 0, rows, cols, inners, blocksize, d_A, d_B, d_C, h_C, h_C_cpu);


		free(h_A);
		free(h_B);
		free(h_C);
		free(h_C_cpu);
		hipFree(d_A);
		hipFree(d_B);
		hipFree(d_C);
		return 0;
}

/*
		case 1: {
			for (int i = 0; i < runs; i++) {
				START
				esmm_shmem_multi<<<dim3(CEIL_DIV(rows, blocksize), CEIL_DIV(cols, blocksize)), dim3(blocksize), blocksize * blocksize * 2 * sizeof(float)>>>(rows, cols, inners, blocksize, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
				hipMemset(d_C, 0, rows * cols * sizeof(float));
			}
			RESULTS("Multi")
			break;
		}
		case 2: {
			for (int i = 0; i < runs; i++) {
				START
				esmm_shmem_multi2<<<dim3(CEIL_DIV(rows, blocksize), CEIL_DIV(cols, blocksize)), dim3(blocksize), blocksize * blocksize * 2 * sizeof(float)>>>(rows, cols, inners, blocksize, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
				hipMemset(d_C, 0, rows * cols * sizeof(float));
				}
			RESULTS("Multi2")
			break;
		}
		case 3: {
			for (int i = 0; i < runs; i++) {
				START
				esmm_shmem_multi3<<<dim3(CEIL_DIV(rows, blocksize), CEIL_DIV(cols, blocksize)), dim3(blocksize), blocksize * blocksize * 2 * sizeof(float)>>>(rows, cols, inners, blocksize, d_A, d_B, d_C);
				END
				hipDeviceSynchronize();
				hipMemset(d_C, 0, rows * cols * sizeof(float));
			}
			RESULTS("Multi3")
			break;
		}
collect_data(runs, 1, rows, cols, inners, blocksize, d_A, d_B, d_C);
			collect_data(runs, 2, rows, cols, inners, blocksize, d_A, d_B, d_C);
			collect_data(runs, 3, rows, cols, inners, blocksize, d_A, d_B, d_C);

*/
