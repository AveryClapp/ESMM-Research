#include <hip/hip_runtime.h>
#include <vector>
#include <random>
#include <algorithm>
#include <numeric>
#include <iostream>

__global__ void matMul(const float* A, const float* B, float* C, int N) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < N && col < N) {
    float sum = 0.0f;
    for (int k = 0; k < N; ++k) {
      sum += A[row * N + k] * B[k * N + col];
    }
    C[row * N + col] = sum;
  }
}

__global__ void matMulEarly(float* C, float* A, float* B, int N, int* colLengths) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < N && col < N) {
    float sum = 0.0f;
    int length = colLengths[col];

    for (int k = 0; k < length; k++) {
      sum += A[row * N + k] * B[k * N + col];
    }
    C[row * N + col] = sum;
  }
}

std::vector<int> generateURVLengths(int N) {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<> dis(0.1, 1.0);

  std::vector<int> lengths(N);
  for (int i = 0; i < N; i++) {
    lengths[i] = static_cast<int>(N * dis(gen));
  }

  return lengths;
}

int main() {
  const int N = 1024;
  size_t bytes = N * N * sizeof(float);

  std::vector<float> h_A(N * N);
  std::vector<float> h_B(N * N);
  std::vector<float> h_C1(N * N);
  std::vector<float> h_C2(N * N);

  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<float> dis(-1.0, 1.0);

  for (int i = 0; i < N * N; i++) {
    h_A[i] = dis(gen);
    h_B[i] = dis(gen);
  }

  std::vector<int> colLengths = generateURVLengths(N);

  std::vector<int> indices(N);
  std::iota(indices.begin(), indices.end(), 0);

  std::sort(indices.begin(), indices.end(),
      [&colLengths](int i1, int i2) { 
      return colLengths[i1] < colLengths[i2]; 
    });

  std::vector<float> h_A_sorted(N * N);
  std::vector<float> h_B_sorted(N * N);
  std::vector<int> sortedLengths(N);

  for(int i = 0; i < N; i++) {
    sortedLengths[i] = colLengths[indices[i]];
    for(int j = 0; j < N; j++) {
      h_B_sorted[j * N + i] = h_B[j * N + indices[i]];
      h_A_sorted[j * N + i] = h_A[j * N + i];
    }
  }

  float *d_A, *d_B, *d_C;
  int *d_lengths;
  hipFree(0); // Initialize context
  hipMalloc(&d_A, bytes);
  hipMalloc(&d_B, bytes);
  hipMalloc(&d_C, bytes);
  hipMalloc(&d_lengths, N * sizeof(int));

  hipMemcpy(d_A, h_A.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B.data(), bytes, hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  dim3 blockDim(32, 32);
  dim3 gridDim((N + blockDim.x - 1) / blockDim.x,
      (N + blockDim.y - 1) / blockDim.y);

  hipEventRecord(start);
  matMul<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float fullTime;
  hipEventElapsedTime(&fullTime, start, stop);

  hipMemcpy(d_A, h_A_sorted.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B_sorted.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_lengths, sortedLengths.data(), N * sizeof(int), hipMemcpyHostToDevice);

  hipEventRecord(start);
  matMulEarly<<<gridDim, blockDim>>>(d_C, d_A, d_B, N, d_lengths);
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float earlyTime;
  hipEventElapsedTime(&earlyTime, start, stop);

  double avgLength = std::accumulate(colLengths.begin(), colLengths.end(), 0.0) / N;
  double maxLength = *std::max_element(colLengths.begin(), colLengths.end());

  std::cout << "Results for N = " << N << ":\n";
  std::cout << "Average column length: " << avgLength << " ("
    << (avgLength/N)*100 << "%)\n";
  std::cout << "Max column length: " << maxLength << " ("
    << (maxLength/N)*100 << "%)\n";
  std::cout << "Full computation time: " << fullTime << " ms\n";
  std::cout << "Early termination time: " << earlyTime << " ms\n";
  std::cout << "Speedup: " << fullTime/earlyTime << "x\n";

  // Cleanup
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  hipFree(d_lengths);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
