#include "hip/hip_runtime.h"
#pragma once

/* Kernel #12, Warptiling (break blocks down even further by controlling warps) */

#include "utils.cuh"
#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <hipblas.h>
#include <hip/hip_runtime.h>

//#include <unrolled_kernels.cuh>

/*
 * @tparam BM The threadblock size for M dimension SMEM caching.
 * @tparam BN The threadblock size for N dimension SMEM caching.
 * @tparam BK The threadblock size for K dimension SMEM caching.
 * @tparam WM M dim of continuous tile computed by each warp
 * @tparam WN N dim of continuous tile computed by each warp
 * @tparam WMITER The number of subwarp tiling steps in M dimension.
 * @tparam WNITER The number of subwarp tiling steps in N dimension.
 * @tparam TM The per-thread tile size for M dimension.
 * @tparam TN The per-thread tile size for N dimension.
 */
template <const int BM, const int BN, const int BK, const int WM, const int WN,
		const int WNITER, const int TM, const int TN, const int NUM_THREADS, const int SIZE>
__global__ void __launch_bounds__(NUM_THREADS)
	esmm_offsets(int M, int N, int K, float *A, float *B, float *C, int* sparse_data) {
	const uint cRow = blockIdx.y;
	const uint cCol = blockIdx.x;

	const uint warpIdx = threadIdx.x / WARPSIZE;
	const uint warpCol = warpIdx % (BN / WN);
	const uint warpRow = warpIdx / (BN / WN);

	constexpr uint WMITER = (WM * WN) / (WARPSIZE * TM * TN * WNITER);
	constexpr uint WSUBM = WM / WMITER;
	constexpr uint WSUBN = WN / WNITER; 

	const uint threadIdxInWarp = threadIdx.x % WARPSIZE;
	const uint threadColInWarp = threadIdxInWarp % (WSUBN / TN); 
	const uint threadRowInWarp = threadIdxInWarp / (WSUBN / TN); 

	__shared__ float As[BN * BK];
	__shared__ float Bs[BM * BK];

	A += cRow * BM * K;
	B += cCol * BN;
	C += (cRow * BM + warpRow * WM) * N + cCol * BN + warpCol * WN;

	const uint innerRowA = threadIdx.x / (BK / 4);
	const uint innerColA = threadIdx.x % (BK / 4);
	constexpr uint rowStrideA = (NUM_THREADS * 4) / BK;
	const uint innerRowB = threadIdx.x / (BN / 4);
	const uint innerColB = threadIdx.x % (BN / 4);
	constexpr uint rowStrideB = NUM_THREADS / (BN / 4);

	float threadResults[WMITER * TM * WNITER * TN] = {0.0};
	float regM[WMITER * TM] = {0.0};
	float regN[WNITER * TN] = {0.0};

	for (int32_t bkIdx = 0; bkIdx < K; bkIdx += BK) {
		for (int32_t offset = 0; offset + rowStrideA <= BM; offset += rowStrideA) {

			const float4 tmp = reinterpret_cast<const float4 *>(
				&A[(innerRowA + offset) * K + innerColA * 4])[0];
			As[(innerColA * 4 + 0) * BM + innerRowA + offset] = tmp.x;
			As[(innerColA * 4 + 1) * BM + innerRowA + offset] = tmp.y;
			As[(innerColA * 4 + 2) * BM + innerRowA + offset] = tmp.z;
			As[(innerColA * 4 + 3) * BM + innerRowA + offset] = tmp.w;
		}
		for (int8_t offset = 0; offset + rowStrideB <= BK; offset += rowStrideB) {
			reinterpret_cast<float4 *>(
				&Bs[(innerRowB + offset) * BN + innerColB * 4])[0] =
				reinterpret_cast<const float4 *>(
					&B[(innerRowB + offset) * N + innerColB * 4])[0];
		}
		__syncthreads();

		#pragma unroll
		for (int sparse_idx = 0; sparse_idx < SIZE; ++sparse_idx) {
			#pragma unroll
			for (int i = 0; i < 2; ++i) { 
				int dotIdx = sparse_data[sparse_idx] + (8 * i);
				for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
					regM[wSubRowIdx] = As[(dotIdx * BM) + warpRow * WM +
						wSubRowIdx * WSUBM + threadRowInWarp * TM];
				}
				for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
					regN[wSubColIdx * TN + 0] = Bs[(dotIdx * BN) + warpCol * 
						WN + wSubColIdx * WSUBN + threadColInWarp * TN + 0];
					regN[wSubColIdx * TN + 1] = Bs[(dotIdx * BN) + warpCol * 
						WN + wSubColIdx * WSUBN + threadColInWarp * TN + 1];
					regN[wSubColIdx * TN + 2] = Bs[(dotIdx * BN) + warpCol * 
						WN + wSubColIdx * WSUBN + threadColInWarp * TN + 2];
					regN[wSubColIdx * TN + 3] = Bs[(dotIdx * BN) + warpCol * 
						WN + wSubColIdx * WSUBN + threadColInWarp * TN + 3];
					regN[wSubColIdx * TN + 4] = Bs[(dotIdx * BN) + warpCol * 
						WN + wSubColIdx * WSUBN + threadColInWarp * TN + 4];
					regN[wSubColIdx * TN + 5] = Bs[(dotIdx * BN) + warpCol * 
						WN + wSubColIdx * WSUBN + threadColInWarp * TN + 5];
					regN[wSubColIdx * TN + 6] = Bs[(dotIdx * BN) + warpCol * 
						WN + wSubColIdx * WSUBN + threadColInWarp * TN + 6];
					regN[wSubColIdx * TN + 7] = Bs[(dotIdx * BN) + warpCol * 
						WN + wSubColIdx * WSUBN + threadColInWarp * TN + 7];
				}
				#pragma unroll
				for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
					#pragma unrol
					for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
						/* switch_table here*/
						multiply_dense(wSubRowIdx, wSubColIdx, WNITER,
						regM[wSubRowIdx], regN, threadResults);
					}
				}
			}
		}
		A += BK;
		B += BK * N;
		__syncthreads();
	}

	for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
		for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
			float *C_interim = C + (wSubRowIdx * WSUBM) * N + wSubColIdx * WSUBN;
			for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
				for (uint resIdxN = 0; resIdxN < TN; resIdxN += 4) {
					float4 tmp;
					const int i = (wSubRowIdx * TM + resIdxM) * (WNITER * TN) +
						wSubColIdx * TN + resIdxN;
					tmp.x = threadResults[i + 0];
					tmp.y = threadResults[i + 1];
					tmp.z = threadResults[i + 2];
					tmp.w = threadResults[i + 3];
					reinterpret_cast<float4 *>(
						&C_interim[(threadRowInWarp * TM + resIdxM) * N +
						threadColInWarp * TN + resIdxN])[0] = tmp;
				}
			}
		}	
	}		
}

