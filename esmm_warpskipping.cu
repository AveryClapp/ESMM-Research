#include "hip/hip_runtime.h"
#pragma once

/* Experimental imporvement to skip warps entirely given a precomputed sparsity */

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"


/*
 * @tparam BM The threadblock size for M dimension SMEM caching.
 * @tparam BN The threadblock size for N dimension SMEM caching.
 * @tparam BK The threadblock size for K dimension SMEM caching.
 * @tparam WM M dim of continuous tile computed by each warp
 * @tparam WN N dim of continuous tile computed by each warp
 * @tparam WMITER The number of subwarp tiling steps in M dimension.
 * @tparam WNITER The number of subwarp tiling steps in N dimension.
 * @tparam TM The per-thread tile size for M dimension.
 * @tparam TN The per-thread tile size for N dimension.
 */
template <const int BM, const int BN, const int BK, const int WM, const int WN,
		const int WNITER, const int TM, const int TN, const int NUM_THREADS>
__global__ void __launch_bounds__(NUM_THREADS)
	esmm_warpskipping(int M, int N, int K, float *A, float *B, float *C) {
	const uint cRow = blockIdx.y;
	const uint cCol = blockIdx.x;

	const uint warpIdx = threadIdx.x / WARPSIZE;
	const uint warpCol = warpIdx % (BN / WN);
	const uint warpRow = warpIdx / (BN / WN);

	constexpr uint WMITER = (WM * WN) / (WARPSIZE * TM * TN * WNITER);
	constexpr uint WSUBM = WM / WMITER;
	constexpr uint WSUBN = WN / WNITER; 

	const uint threadIdxInWarp = threadIdx.x % WARPSIZE;
	/* This ends up being 0 every time so each thread starts at col 0 */
	const uint threadColInWarp = threadIdxInWarp % (WSUBN / TN); 
	/* This ends up being threadIdxInWarp so you assign rows sequentially. */
	const uint threadRowInWarp = threadIdxInWarp / (WSUBN / TN); 

	__shared__ float As[BN * BK];
	__shared__ float Bs[BM * BK];

	A += cRow * BM * K;
	B += cCol * BN;
	C += (cRow * BM + warpRow * WM) * N + cCol * BN + warpCol * WN;

	const uint innerRowA = threadIdx.x / (BK / 4);
	const uint innerColA = threadIdx.x % (BK / 4);
	constexpr uint rowStrideA = (NUM_THREADS * 4) / BK;
	const uint innerRowB = threadIdx.x / (BN / 4);
	const uint innerColB = threadIdx.x % (BN / 4);
	constexpr uint rowStrideB = NUM_THREADS / (BN / 4);


	float threadResults[WMITER * TM * WNITER * TN] = {0.0};
	//float regM[WMITER * TM] = {0.0};
	float regN[WNITER * TN] = {0.0};

	for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
		for (uint offset = 0; offset + rowStrideA <= BM; offset += rowStrideA) {
			const float4 tmp = reinterpret_cast<const float4 *>(
				&A[(innerRowA + offset) * K + innerColA * 4])[0];
				As[(innerColA * 4 + 0) * BM + innerRowA + offset] = tmp.x;
		  		As[(innerColA * 4 + 1) * BM + innerRowA + offset] = tmp.y;
			  	As[(innerColA * 4 + 2) * BM + innerRowA + offset] = tmp.z;
				As[(innerColA * 4 + 3) * BM + innerRowA + offset] = tmp.w;
		}
		for (uint offset = 0; offset + rowStrideB <= BK; offset += rowStrideB) {
			reinterpret_cast<float4 *>( 
				&Bs[(innerRowB + offset) * BN + innerColB * 4])[0] =
			reinterpret_cast<const float4 *>(
				&B[(innerRowB + offset) * N + innerColB * 4])[0];
		}
		__syncthreads();
		for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
			for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
				for (uint inner = 0; inner < WSUBM; ++inner) {
					/* Have the first thread load its A-value */
					float cur_val = 0.0f;
					if ((threadIdx.x & 31) == 0) {
						cur_val = As[dotIdx * BM + warpRow * WM + wSubRowIdx * WSUBM
							+ threadRowInWarp + inner];
					}
					unsigned active_threads = __activemask();
					/* Broadcast the A-value to all other threads in the warp */
					float a_val = __shfl_sync(active_threads, cur_val, 0);
					/* All threads will skip if the A-value is 0 for thread 0 */
					if (a_val == 0)
						continue;
					for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
						for (uint i = 0; i < TN; ++i) {
							regN[wSubColIdx * TN + i] = 
								Bs[dotIdx * BN + warpCol * WN + wSubColIdx 
									* WSUBN + threadColInWarp * TN + i];
						}
						multiply_dense(wSubRowIdx, wSubColIdx, WNITER, 
										a_val, regN, threadResults);
					}
				}
			}
		}
		A += BK;     
		B += BK * N; 
		__syncthreads();
	}
	for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
		for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
			float *C_interim = C + (wSubRowIdx * WSUBM) * N + wSubColIdx * WSUBN;
				for (uint resIdxM = 0; resIdxM < TM; resIdxM += 1) {
					for (uint resIdxN = 0; resIdxN < TN; resIdxN += 4) {
						float4 tmp;
						const int i = (wSubRowIdx * TM + resIdxM) * (WNITER * TN) +
							wSubColIdx * TN + resIdxN;
							tmp.x = threadResults[i + 0];
							tmp.y = threadResults[i + 1];
							tmp.z = threadResults[i + 2];
							tmp.w = threadResults[i + 3];
							reinterpret_cast<float4 *>(
								&C_interim[(threadRowInWarp * TM + resIdxM) * N +
									threadColInWarp * TN + resIdxN])[0] = tmp;
					}
				}
		}	
	}		
}

