#include "hip/hip_runtime.h"
#pragma once
#include "utils.cuh"
#include <hip/hip_runtime.h>

template <const int BM, const int BN, const int BK, const int WM, const int WN,
    const int WNITER, const int TM, const int TN, const int NUM_THREADS>
__global__ void __launch_bounds__(NUM_THREADS)
    esmm_workqueue(int M, int N, int K, float *A, float *B, float *C) {
        const uint cRow = blockIdx.y;
        const uint cCol = blockIdx.x;

        const uint warpIdx = threadIdx.x / WARPSIZE;
        const uint warpCol = warpIdx % (BN / WN);
        const uint warpRow = warpIdx / (BN / WN);

        constexpr uint WMITER = (WM * WN) / (WARPSIZE * TM * TN * WNITER);
        constexpr uint WSUBM = WM / WMITER;
        constexpr uint WSUBN = WN / WNITER;

        const uint threadIdxInWarp = threadIdx.x % WARPSIZE;
        const uint threadColInWarp = threadIdxInWarp % (WSUBN / TN);
        const uint threadRowInWarp = threadIdxInWarp / (WSUBN / TN);

        __shared__ float As[BM * BK];
        __shared__ float Bs[BK * BN];

        constexpr uint NUM_WARPS = NUM_THREADS / WARPSIZE;
        __shared__ struct {
            uint offset[WMITER * WARPSIZE];
            float value[WMITER * WARPSIZE];
            uint count;
        } warpQueues[NUM_WARPS];

        A += cRow * BM * K;
        B += cCol * BN;
        C += (cRow * BM + warpRow * WM) * N + cCol * BN + warpCol * WN;

        const uint innerRowA = threadIdx.x / (BK / 4);
        const uint innerColA = threadIdx.x % (BK / 4);
        constexpr uint rowStrideA = (NUM_THREADS * 4) / BK;
        const uint innerRowB = threadIdx.x / (BN / 4);
        const uint innerColB = threadIdx.x % (BN / 4);
        constexpr uint rowStrideB = NUM_THREADS / (BN / 4);

        float threadResults[WMITER * TM * WNITER * TN] = {0.0};
        float regM[WMITER * TM] = {0.0};
        float regN[WNITER * TN] = {0.0};

        for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
            for (uint offset = 0; offset + rowStrideA <= BM; offset += rowStrideA) {
                const float4 tmp = reinterpret_cast<const float4 *>(
                        &A[(innerRowA + offset) * K + innerColA * 4])[0];
                As[(innerColA * 4 + 0) * BM + innerRowA + offset] = tmp.x;
                As[(innerColA * 4 + 1) * BM + innerRowA + offset] = tmp.y;
                As[(innerColA * 4 + 2) * BM + innerRowA + offset] = tmp.z;
                As[(innerColA * 4 + 3) * BM + innerRowA + offset] = tmp.w;
            }
            for (uint offset = 0; offset + rowStrideB <= BK; offset += rowStrideB) {
                reinterpret_cast<float4 *>(
                        &Bs[(innerRowB + offset) * BN + innerColB * 4])[0] =
                    reinterpret_cast<const float4 *>(
                            &B[(innerRowB + offset) * N + innerColB * 4])[0];
            }
            __syncthreads();

            for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
                if (threadIdxInWarp == 0) {
                    warpQueues[warpIdx].count = 0;
                }
                __syncwarp();

                for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
                    float val = As[(dotIdx * BM) + warpRow * WM + 
                        wSubRowIdx * WSUBM + threadRowInWarp * TM];
                    regM[wSubRowIdx] = val;

                    if (val != 0) {
                        uint idx = atomicAdd(&warpQueues[warpIdx].count, 1);
                        warpQueues[warpIdx].offset[idx] = wSubRowIdx;
                        warpQueues[warpIdx].value[idx] = val;
                    }
                }
                __syncwarp();

                for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
                    regN[wSubColIdx * TN + 0] = Bs[(dotIdx * BN) + warpCol * WN + 
                        wSubColIdx * WSUBN + threadColInWarp * TN + 0];
                    regN[wSubColIdx * TN + 1] = Bs[(dotIdx * BN) + warpCol * WN + 
                        wSubColIdx * WSUBN + threadColInWarp * TN + 1];
                    regN[wSubColIdx * TN + 2] = Bs[(dotIdx * BN) + warpCol * WN + 
                        wSubColIdx * WSUBN + threadColInWarp * TN + 2];
                    regN[wSubColIdx * TN + 3] = Bs[(dotIdx * BN) + warpCol * WN + 
                        wSubColIdx * WSUBN + threadColInWarp * TN + 3];
                    regN[wSubColIdx * TN + 4] = Bs[(dotIdx * BN) + warpCol * WN + 
                        wSubColIdx * WSUBN + threadColInWarp * TN + 4];
                    regN[wSubColIdx * TN + 5] = Bs[(dotIdx * BN) + warpCol * WN + 
                        wSubColIdx * WSUBN + threadColInWarp * TN + 5];
                    regN[wSubColIdx * TN + 6] = Bs[(dotIdx * BN) + warpCol * WN + 
                        wSubColIdx * WSUBN + threadColInWarp * TN + 6];
                    regN[wSubColIdx * TN + 7] = Bs[(dotIdx * BN) + warpCol * WN + 
                        wSubColIdx * WSUBN + threadColInWarp * TN + 7];
                }

                uint totalWork = warpQueues[warpIdx].count;
                for (uint i = threadIdxInWarp; i < totalWork; i += WARPSIZE) {
                    uint wSubRowIdx = warpQueues[warpIdx].offset[i];
                    float aVal = warpQueues[warpIdx].value[i];
                    for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
                        multiply_dense(wSubRowIdx, wSubColIdx, WNITER, aVal, regN, threadResults);
                    }
                }
            }
            A += BK;
            B += BK * N;
            __syncthreads();
        }

        for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
            for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
                float *C_interim = C + (wSubRowIdx * WSUBM) * N + wSubColIdx * WSUBN;
                for (uint resIdxM = 0; resIdxM < TM; resIdxM += 1) {
                    for (uint resIdxN = 0; resIdxN < TN; resIdxN += 4) {
                        float4 tmp;
                        const int i = (wSubRowIdx * TM + resIdxM) * (WNITER * TN) +
                            wSubColIdx * TN + resIdxN;
                        tmp.x = threadResults[i + 0];
                        tmp.y = threadResults[i + 1];
                        tmp.z = threadResults[i + 2];
                        t
