#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "utils.cuh"
#include <iostream>


__global__ void matrixMultiply(const float* A, const float* B, float* C) {
  int row = blockIdx.x * 32 + threadIdx.x;      
  float aVal = A[row * 32 + threadIdx.x];
  for(int j = 0; j < 8; j++) {
    C[row * 8 + j] += aVal * B[threadIdx.x * 8 + j];
  }
}

int main() {
  float *A, *B, *C;
  float *d_A, *d_B, *d_C;
  
  hipFree(0);
  A = (float*)malloc(1024 * 32 * sizeof(float));
  B = (float*)malloc(32 * 8 * sizeof(float));
  C = (float*)malloc(1024 * 8 * sizeof(float));

  randomize_matrix(A, 1024, 32);
  randomize_matrix(B, 32, 8);

  hipMalloc(&d_A, 1024 * 32 * sizeof(float));
  hipMalloc(&d_B, 32 * 8 * sizeof(float));
  hipMalloc(&d_C, 1024 * 8 * sizeof(float));

  hipMemcpy(d_A, A, 1024 * 32 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, 32 * 8 * sizeof(float), hipMemcpyHostToDevice);

  dim3 threadsPerBlock(32, 1);
  dim3 numBlocks(32, 1);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  matrixMultiply<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);
  hipEventRecord(stop);

  hipMemcpy(C, d_C, 1024 * 8 * sizeof(float), hipMemcpyDeviceToHost);
  hipEventSynchronize(stop);
  float time = 0.0f;
  hipEventElapsedTime(&time, start, stop);

  std::cout << "GPU Timing: " << time << " ms" << std::endl;
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(A);
  free(B);
  free(C);

  return 0;
}
