#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include "utils.cuh"
#include <iostream>

#define A_ROWS 1024
#define A_COLS 32
#define B_ROWS 32
#define B_COLS 8
#define C_ROWS 1024
#define C_COLS 8

__global__ void matrixMultiply(const float* A, const float* B, float* C) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
      if (row < 1024) {
            for (int j = 0; j < 8; j++) {
                    float sum = 0.0f;
                          for (int k = 0; k < 32; k++) {
                                    sum += A[row * 32 + k] * B[k * 8 + j];
                                          }
                                C[row * 8 + j] = sum;
                                    }
              }

}
int main() {
  float *A, *B, *C, *C_cpu;
  float *d_A, *d_B, *d_C;

  hipFree(0);
  A = (float*)malloc(1024 * 32 * sizeof(float));
  B = (float*)malloc(32 * 8 * sizeof(float));
  C = (float*)malloc(1024 * 8 * sizeof(float));
  C_cpu = (float*)malloc(1024 * 8 * sizeof(float));

  randomize_matrix(A, 1024, 32);
  randomize_matrix(B, 32, 8);

  hipMalloc(&d_A, 1024 * 32 * sizeof(float));
  hipMalloc(&d_B, 32 * 8 * sizeof(float));
  hipMalloc(&d_C, 1024 * 8 * sizeof(float));

  hipMemcpy(d_A, A, 1024 * 32 * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, 32 * 8 * sizeof(float), hipMemcpyHostToDevice);
  
  int threadsPerBlock = 32;
  int blocksPerGrid = (1024 + threadsPerBlock - 1) / threadsPerBlock;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);
  hipEventRecord(stop);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch matrixMultiply kernel (error code %s)!\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  hipEventSynchronize(stop);
  hipMemcpy(C, d_C, 1024 * 8 * sizeof(float), hipMemcpyDeviceToHost);
  float time = 0.0f;
  hipEventElapsedTime(&time, start, stop);

  for (int i = 0; i < 1024; i++) {
    for (int j = 0; j < 8; j++) {
      C_cpu[i * 8 + j] = 0.0f; // Initialize C_cpu[i][j] to zero
      for (int k = 0; k < 32; k++) {
        C_cpu[i * 8 + j] += A[i * 32 + k] * B[k * 8 + j];
      }
    }
  }

  bool correct = true;
  for (int i = 0; i < 1024; i++) {
    for (int j = 0; j < 8; j++) {
      if (fabs(C[i * 8 + j] - C_cpu[i * 8 + j]) > 1e-4) { 
        std::cout << C[i * 8 + j] << C_cpu[i * 8 + j] << std::endl;
        correct = false; // Matrices are not equal
        break; // Exit the inner loop if a discrepancy is found
      }
    }
    if (!correct) break; // Exit the outer loop if a discrepancy is found
  }

  if (correct) {
    std::cout << "Matrix multiplication is correct!" << std::endl;
  } else {
    std::cout << "Matrix multiplication is incorrect!" << std::endl;
  }

  std::cout << "GPU Timing: " << time << " ms" << std::endl;
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  free(A);
  free(B);
  free(C);

  return 0;
}
