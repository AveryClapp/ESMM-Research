#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"
// Error checking macro
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}

#define A_ROWS 1024
#define A_COLS 32
#define B_ROWS 32
#define B_COLS 8
#define C_ROWS 1024
#define C_COLS 8

__global__ void matrixMultiplyKernel(float* A, float* B, float* C, int bTileSize, int aTileSize) {
  // Each thread handles one row of B
  int row = (blockIdx.y * blockDim.x + blockIdx.x) + (blockIdx.z * aTileSize);
  int b_col = bTileSize * threadIdx.y;
  int b_row = threadIdx.x;
  if (b_row < B_ROWS) {
    // Store all the elements that this thread will process (next 4 elements (inclusive))
    float thread_elements[8] = {
      B[b_row * B_COLS + (b_col * bTileSize) + 0],
      B[b_row * B_COLS + (b_col * bTileSize) + 1],
      B[b_row * B_COLS + (b_col * bTileSize) + 2],
      B[b_row * B_COLS + (b_col * bTileSize) + 3],
    };
    float a_element = A[row * A_COLS + b_row];
    atomicAdd(&C[row * C_COLS + 0], a_element * thread_elements[0]);
    atomicAdd(&C[row * C_COLS + 1], a_element * thread_elements[1]);
    atomicAdd(&C[row * C_COLS + 2], a_element * thread_elements[2]);
    atomicAdd(&C[row * C_COLS + 3], a_element * thread_elements[3]);
  }
}

void matrixMultiplyCPU(float* A, float* B, float* C) {
  for (int row = 0; row < A_ROWS; row++) {
    for (int col = 0; col < B_COLS; col++) {
      float sum = 0.0f;
      for (int i = 0; i < A_COLS; i++) {
        sum += A[row * A_COLS + i] * B[i * B_COLS + col];
      }
      C[row * C_COLS + col] = sum;
    }
  }
}

// Verify results
bool verifyResults(float* gpuResult, float* cpuResult, int size, float tolerance = 1e-5) {
  for (int i = 0; i < size; i++) {
    if (fabs(gpuResult[i] - cpuResult[i]) > tolerance) {
      printf("Mismatch at position %d: GPU = %f, CPU = %f\n", 
          i, gpuResult[i], cpuResult[i]);
      return false;
    }
  }
  return true;
}

int main() {
  float *h_A = (float*)malloc(A_ROWS * A_COLS * sizeof(float));
  float *h_B = (float*)malloc(B_ROWS * B_COLS * sizeof(float));
  float *h_C = (float*)malloc(C_ROWS * C_COLS * sizeof(float));
  float *h_C_cpu = (float*)malloc(C_ROWS * C_COLS * sizeof(float));

  randomize_matrix(h_A, A_ROWS, A_COLS);
  randomize_matrix(h_B, B_ROWS, B_COLS);

  float *d_A, *d_B, *d_C;
  cudaCheckError(hipMalloc(&d_A, A_ROWS * A_COLS * sizeof(float)));
  cudaCheckError(hipMalloc(&d_B, B_ROWS * B_COLS * sizeof(float)));
  cudaCheckError(hipMalloc(&d_C, C_ROWS * C_COLS * sizeof(float)));

  cudaCheckError(hipMemcpy(d_A, h_A, A_ROWS * A_COLS * sizeof(float), hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(d_B, h_B, B_ROWS * B_COLS * sizeof(float), hipMemcpyHostToDevice));

  hipFree(0);
  hipMemset(d_C, 0, C_ROWS * C_COLS * sizeof(float));

  dim3 gridDim(32,16,2);
  dim3 blockDim(32,2);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  matrixMultiplyKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C,4,512);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  cudaCheckError(hipMemcpy(h_C, d_C, C_ROWS * C_COLS * sizeof(float), hipMemcpyDeviceToHost));

  float time = 0.0f;
  hipEventElapsedTime(&time, start, stop);

  std::cout << "GPU Timing: " << time << " ms" << std::endl;
  matrixMultiplyCPU(h_A, h_B, h_C_cpu);

  bool correct = verifyResults(h_C, h_C_cpu, C_ROWS * C_COLS);
  printf("Matrix multiplication %s\n", correct ? "PASSED" : "FAILED");

  free(h_A);
  free(h_B);
  free(h_C);
  free(h_C_cpu);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return 0;
}
