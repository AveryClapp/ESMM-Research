#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"


/* ------------------ OLD KERNELS ------------------------- */
// Original Method for a 32x32 block of B -- OUT OF DATE
__global__ void basic32(float* A, float* B, float* C) {
	// Each thread handles one row of B
	int b_row = threadIdx.x;
	int row = blockIdx.y * blockDim.x + blockIdx.x;
	if (b_row < B_ROWS) {
		float a_element = A[row * A_COLS + b_row];
		for (int j = 0; j < B_COLS; j++) {
			int b_col = (threadIdx.x + j) % B_COLS;
			C[blockIdx.x * C_COLS + b_col] += a_element * B[b_row * B_COLS + b_col];
		}
	}
}

// Original Method for a 8x8 block of B, impossible to eleminate atomicAdd
// here (pigeonhole principle)
__global__ void basic8(float* A, float* B, float* C) {
	int b_row = threadIdx.x;
	if (b_row < B_ROWS) {
		float b_elements[8] = {
			B[b_row * B_COLS + 0],
			B[b_row * B_COLS + 1],
			B[b_row * B_COLS + 2],
			B[b_row * B_COLS + 3],
			B[b_row * B_COLS + 4],
			B[b_row * B_COLS + 5],
			B[b_row * B_COLS + 6],
			B[b_row * B_COLS + 7]
		};
		float a_element = A[blockIdx.x * A_COLS + b_row];
		atomicAdd(&C[blockIdx.x * C_COLS + 0], a_element * b_elements[0]);
		atomicAdd(&C[blockIdx.x * C_COLS + 1], a_element * b_elements[1]);
		atomicAdd(&C[blockIdx.x * C_COLS + 2], a_element * b_elements[2]);
		atomicAdd(&C[blockIdx.x * C_COLS + 3], a_element * b_elements[3]);
		atomicAdd(&C[blockIdx.x * C_COLS + 4], a_element * b_elements[4]);
		atomicAdd(&C[blockIdx.x * C_COLS + 5], a_element * b_elements[5]);
		atomicAdd(&C[blockIdx.x * C_COLS + 6], a_element * b_elements[6]);
		atomicAdd(&C[blockIdx.x * C_COLS + 7], a_element * b_elements[7]);
	}
}

/* ---------------------- NEW KERNELS --------------------------------------- */


template <const int blockHeight, const int blockWidth, const int blockInner, const int TM>
__global__ void matMulBlocktiling(float* A, float* B, float* C, int N, int innerDim) {
	const int col = blockIdx.x;
	const int row = blockIdx.y;

	// Designated col and row for each thread
	const int localCol = threadIdx.x % blockWidth; // Assign cols sequentially
	const int localRow = threadIdx.x / blockWidth; // Assign rows in batches

	A += row * innerDim * blockHeight; 
	B += col * blockWidth;
	C += row * innerDim * blockHeight + col * blockWidth;

	float threadResults[TM] = {0.0};

	//Iterate over column/row
	//128 % 8 == 0, so this is fine for the time being
	for (int blockIndex = 0; blockIndex < innerDim; blockIndex += blockInner) {
	    __syncthreads();
		
		A += blockInner;
		B += blockInner * N;

		for (int dotIdx = 0; dotIdx < blockInner; ++dotIdx) {
			float tmpB = B[dotIdx * blockWidth + localCol];
			for (uint resIdx = 0; resIdx < TM; ++resIdx) {
				threadResults[resIdx] += 
					A[(localRow * TM + resIdx) * blockInner + dotIdx] * tmpB;
			}
		}
		__syncthreads();
	}
	for (int resIdx = 0; resIdx < TM; ++resIdx) {
		C[(localRow * TM + resIdx) * N + localCol] = threadResults[resIdx];
	}
}


template <const int blockHeight, const int blockWidth, const int blockInner, const int TM>
__global__ void matMulBlocktilingTwo(float* A, float* B, float* C, int N, int innerDim) {
		const int col = blockIdx.x;
		const int row = blockIdx.y;

		// Thread indexing remains the same
		const int localCol = threadIdx.x % blockWidth;
		const int localRow = threadIdx.x / blockWidth;

		// Store base pointers
		float* baseA = A + row * innerDim * blockHeight;
		float* baseB = B + col * blockWidth;
		float* baseC = C + row * N * blockHeight + col * blockWidth;

		float threadResults[TM] = {0.0f};

		// Iterate over blocks
		for (int blockIndex = 0; blockIndex < innerDim; blockIndex += blockInner) {
				__syncthreads();

				// Calculate
				// current
				// block
				// pointers
				float* currA = baseA + blockIndex;
				float* currB = baseB + blockIndex * N;

				// Compute
				// dot
				// products
				for (int dotIdx = 0; dotIdx < blockInner; ++dotIdx) {
						float tmpB = currB[dotIdx * N + localCol];  // Fixed
																	// B
																	// indexing

						for (uint resIdx = 0; resIdx < TM; ++resIdx) {
								float tmpA = currA[(localRow * TM + resIdx) * innerDim + dotIdx];  // Fixed
																								   // A
																								   // indexing
								threadResults[resIdx] += tmpA * tmpB;
						}
				}
				__syncthreads();
		}

		// Write results
		for (int resIdx = 0; resIdx < TM; ++resIdx) {
				baseC[(localRow * TM + resIdx) * N + localCol] = threadResults[resIdx];
		}
}
