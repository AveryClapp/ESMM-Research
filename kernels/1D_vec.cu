#pragma once

/* Kernel #6, Transposing A matrix into SMEM for better memory access */

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>


template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void __launch_bounds__((BM * BN) / (TM * TN), 1)
	one_d_vec(int M, int N, int K, float *A,
					 float *B, float *C) {
	// Determines where the block will start
	const uint cRow = blockIdx.y;
	const uint cCol = blockIdx.x;

	// We calculate BM * BN elements per block, must find how many threads are
	// needed total
	const uint totalResultsBlocktile = BM * BN;
	const uint numThreadsBlocktile = totalResultsBlocktile / (TM);

	assert(numThreadsBlocktile == blockDim.x);

	// Blocked groups of cols and sequential rows
	// Assign threadCol and threadRow in row-major order 
	const int threadCol = threadIdx.x % BN;
	const int threadRow = threadIdx.x / BN;

	__shared__ float As[BM * BK];
	__shared__ float Bs[BK * BN];

	// Advance matrix pointers to the start of the block
	A += cRow * BM * K;
	B += cCol * BN;
	C += cRow * BM * N + cCol * BN;

	const uint innerRowA = threadIdx.x / (BK / 4);
	const uint innerColA = threadIdx.x % (BK / 4);
  	const uint innerRowB = threadIdx.x / (BN / 4);
	const uint innerColB = threadIdx.x % (BN / 4);

	float threadResults[TM] = {0.0}; // All thread results

	// Every advance the block through the matrix
	for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
		float4 tmp = reinterpret_cast<const float4 *>(&A[innerRowA * K + innerColA * 4])[0];
		
		// Load elements from row major order in A to column major in As
		As[(innerColA * 4 + 0) * BM + innerRowA] = tmp.x;
   		As[(innerColA * 4 + 1) * BM + innerRowA] = tmp.y;
	    As[(innerColA * 4 + 2) * BM + innerRowA] = tmp.z;
		As[(innerColA * 4 + 3) * BM + innerRowA] = tmp.w;

		// Load the float4 value from global memory
		reinterpret_cast<float4 *>(&Bs[innerRowB * BN + innerColB * 4])[0] = reinterpret_cast<float4 *>(&B[innerRowB * N + innerColB * 4])[0];

		__syncthreads();

		// Advance the matrix pointers to the start of the next block
		A += BK;
		B += BK * N;
			
		for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
			// This goes down the column of B (BK elems)
			float bTmp = Bs[dotIdx * BN + threadCol];

			// Calculate TM elements for every thread
			for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
				threadResults[resIdxM] += As[dotIdx * BM + threadRow * TM + resIdxM] * bTmp;
			}
		}
		__syncthreads();
	}

	// Apply thread results
	for (uint resIdx = 0; resIdx < TM; ++resIdx) {
		C[(threadRow * TM + resIdx) * N + threadCol] = threadResults[resIdx];
	}
}
