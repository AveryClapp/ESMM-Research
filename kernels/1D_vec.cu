#pragma once

/* Kernel #6, Transposing A matrix into SMEM for better memory access */

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>


template <const int BM, const int BN, const int BK, const int TM, const int TN>
__global__ void __launch_bounds__((BM * BN) / (TM * TN), 1)
	one_d_vec(int M, int N, int K, float *A,
					 float *B, float *C) {
	// Determines where the block will start
	const uint cRow = blockIdx.y;
	const uint cCol = blockIdx.x;

	// We calculate BM * BN elements per block, must find how many threads are
	// needed total (including both dimensions)
	const uint totalResultsBlocktile = BM * BN;
	const uint numThreadsBlocktile = totalResultsBlocktile / (TM);

	assert(numThreadsBlocktile == blockDim.x);

	// Blocked groups of cols and sequential rows
	// Assign threadCol and threadRow in row-major order 
	const int threadCol = threadIdx.x % (BN);
	const int threadRow = threadIdx.x / (BN);

	__shared__ float As[BM * BK];
	__shared__ float Bs[BK * BN];

	// Advance matrix pointers to the start of the block
	A += cRow * BM * K;
	B += cCol * BN;
	C += cRow * BM * N + cCol * BN;

	const uint innerRowA = threadIdx.x / (BK / 4);
	const uint innerColA = threadIdx.x % (BK / 4);
  	const uint innerRowB = threadIdx.x / (BN / 4);
	const uint innerColB = threadIdx.x % (BN / 4);

	float threadResults[TM] = {0.0}; // All thread results

	// Middle loop caching
	float regM[TM] = {0.0};
	float regN[TN] = {0.0}; 

	// Every advance the block through the matrix
	for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
		float4 tmp = 
		  reinterpret_cast<const float4 *>(&A[innerRowA * K + innerColA * 4])[0];
		
		// Load elements from row major order in A to column major in As
		As[(innerColA * 4 + 0) * BM + innerRowA] = tmp.x;
   		As[(innerColA * 4 + 1) * BM + innerRowA] = tmp.y;
	    As[(innerColA * 4 + 2) * BM + innerRowA] = tmp.z;
		As[(innerColA * 4 + 3) * BM + innerRowA] = tmp.w;

		// Load the float4 value from global memory
		reinterpret_cast<float4 *>(&Bs[innerRowB * BN + innerColB * 4])[0] =
		  reinterpret_cast<float4 *>(&B[innerRowB * N + innerColB * 4])[0];
		__syncthreads();

		// Advance the matrix pointers to the start of the next block
		A += BK;
		B += BK * N;
			
		for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
			// Store values to be used in inner loop
			for (uint i = 0; i < TM; ++i) {
				regM[i] = As[dotIdx * BM + threadRow * TM + i];
			}

			// Just one element, can simplify this when it starts to work
			for (uint i = 0; i < TN; ++i) {
				regN[i] = Bs[dotIdx * BN + threadCol + i];
			}

			// Calculate TM * TN elements in current block
			for (uint resIdxM = 0; resIdxM < TM; ++resIdxM) {
				for (uint resIdxN = 0; resIdxN < TN; ++resIdxN) {
					threadResults[resIdxM + resIdxN] +=
						regM[resIdxM] * regN[resIdxN];
				}
			}
		}
		__syncthreads();
	}

	for (uint resIdx = 0; resIdx < TM; ++resIdx) {
			    C[(threadRow * TM + resIdx) * N + threadCol] = threadResults[resIdx];
	}
	// Accumulate results from thread results registerfile into C
//	for (uint resIdxM = 0; resIdxM < TM; resIdxM += 1) {
	//	for (uint resIdxN = 0; resIdxN < TN; resIdxN += 4) {
	//		float4 tmp;
	//		tmp.x = threadResults[resIdxM + resIdxN];
	//		tmp.y = threadResults[resIdxM + resIdxN + 1];
	//		tmp.z = threadResults[resIdxM + resIdxN + 2];
	//		tmp.w = threadResults[resIdxM + resIdxN + 3];
	//		reinterpret_cast<float4 *>(&C[(threadRow * TM + resIdxM) * N + threadCol + resIdxN])[0] = tmp;
	//	}
	//}
}
