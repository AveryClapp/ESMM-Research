#include <hip/hip_runtime.h>

#include <iostream>

__global__ void esmm_shmem_multi2 (int rows, int columns, int inners, 
				int blocksize,
				const float *A, const float *B, float *C)
{
	const int row = blockIdx.x * blocksize;
	const int col = blockIdx.y * blocksize + (threadIdx.x % blocksize);

	int coloff = col % blocksize;

	extern __shared__ float sArea [];
	float* sA = sArea;  
	float* sB = sArea + blocksize * blocksize; 

	float tmpres[8] = {0.0}; // thread results

	for (int inner=0; inner < inners; inner += blocksize)
	{
		for (int dotidx=0; dotidx<blocksize; dotidx++)
		{
			sA[dotidx * blocksize + coloff] = A[(row + dotidx) * inners + inner + coloff];
			sB[dotidx * blocksize + coloff] = B[(inner + dotidx) * columns + col];
		}
		__syncthreads();

		for (int dotidx=0; dotidx < blocksize; dotidx++)
		{
			for (int i=0; i < blocksize; ++i)
			{
				tmpres[dotidx] +=  sA[dotidx * blocksize + i] * sB[i * blocksize + coloff];
			}
		}
		__syncthreads();
	}

	for (int dotidx=0; dotidx<blocksize; dotidx++)
	{
		C[(row + dotidx) * columns + col] = tmpres[dotidx];
	}
	return;
}
