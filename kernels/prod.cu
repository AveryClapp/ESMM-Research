#include <hip/hip_runtime.h>

#include <iostream>

__global__ void esmm_shmem_multi (int rows, int columns, int inners, 
				int blocksize,
				const float *A, const float *B, float *C)
{
	const int row = blockIdx.x * blocksize;
	const int col = blockIdx.y * blocksize + (threadIdx.x % blocksize);

	int coloff = col % blocksize;

	extern __shared__ float sArea [];
	float* sA = sArea;  
	float* sB = sArea + blocksize * blocksize; 

	float tmpres[8] = {0.0}; 

	for (int inner=0; inner < inners; inner += blocksize)
	{
		for (int dotidx=0; dotidx<blocksize; dotidx++)
		{
			sA[dotidx * blocksize + coloff] = A[(row + dotidx) * inners + inner + coloff];
			sB[dotidx * blocksize + coloff] = B[(inner + dotidx) * columns + col];
		}
		//__syncthreads();

		for (int i=0; i < blocksize; ++i)
		{
			float Btmp = sB[i * blocksize + coloff];
			for (int dotidx=0; dotidx < blocksize; dotidx++)
			{
				tmpres[dotidx] +=  sA[dotidx * blocksize + i] * Btmp;
			}
		}
			__syncthreads();
	}

	for (int dotidx=0; dotidx<blocksize; dotidx++)
	{
		C[(row + dotidx) * columns + col] = tmpres[dotidx];
	}
	return;
}
