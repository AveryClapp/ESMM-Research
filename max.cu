#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
		    int device = 0;  // Assuming you want to query the first GPU
			    hipSetDevice(device);
				    
				    int sharedMemPerMultiprocessor;
					    hipDeviceGetAttribute(&sharedMemPerMultiprocessor, 
										                           hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, 
																                              device);
						    
						    printf("Maximum Shared Memory Per Multiprocessor: %d bytes (0x%X)\n", 
											           sharedMemPerMultiprocessor, sharedMemPerMultiprocessor);
							    
							int sharedMemPerBlock;
							hipDeviceGetAttribute(&sharedMemPerBlock, 
											                      hipDeviceAttributeSharedMemPerBlockOptin, 
																                        device);
							    
							printf("Maximum Shared Memory Per Block: %d bytes (0x%X)\n", 
											       sharedMemPerBlock, sharedMemPerBlock);
							    return 0;
}
