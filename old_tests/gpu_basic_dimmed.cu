#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"
// Error checking macro
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
    exit(code);
  }
}

#define A_ROWS 1024
#define A_COLS 32
#define B_ROWS 32
#define B_COLS 8
#define C_ROWS 1024
#define C_COLS 8
/*
In order for us not to use atomicadd, we need to make sure that indices in C are being written to one at a time and
there are no race conditions. One way to do this is to have one thread be purely responsible for one index in C, 
which would require the thread to go across a row of A and down a col of B. This would drastically change the 
architecture of the problem we are trying to solve as we would then be launching 8 threads per block instead of 32,
this is inefficient as warp sizes are 32 at the smallest, so we're just wasting the overhead of 24 threads here. 
TLDR: one thread cant compute one index in C. So the ONLY other option is that we need to modify accesses to ensure
that one thread operates on one index of C. Since each kernel launch only considers one row of C, we need to only
consider one kernel (since muktiple kernels won't overlap with eachother as they only write to one row of C).
If this is the case, we need to somehow make acceses such that, 32 threads never access the same index in a row of
8. Hmm.... A thread accesses the index of C based on which column it is on. Pigeonhole principle tells us this is
impossible. 32 threads, 8 columns, impossible. So, as long as threads > columns, you can't eliminate atomic add.
*/
__global__ void matrixMultiplyKernel(float* A, float* B, float* C, int bTileSize, int aTileSize) {
  // Each thread handles one row of B
  int row = (blockIdx.y * blockDim.x + blockIdx.x) + (blockIdx.z * aTileSize);
  int b_row = threadIdx.x;
  if (b_row < B_ROWS) {
    // Store all the elements that this thread will process (next 4 elements (inclusive))
    float thread_elements[8] = {
      B[b_row * B_COLS + 0],
      B[b_row * B_COLS + 1],
      B[b_row * B_COLS + 2],
      B[b_row * B_COLS + 3],
      B[b_row * B_COLS + 4],
      B[b_row * B_COLS + 5],
      B[b_row * B_COLS + 6],
      B[b_row * B_COLS + 7]
    };
    float a_element = A[row * A_COLS + b_row];
    atomicAdd(&C[row * C_COLS + 0], a_element * thread_elements[0]);
    atomicAdd(&C[row * C_COLS + 1], a_element * thread_elements[1]);
    atomicAdd(&C[row * C_COLS + 2], a_element * thread_elements[2]);
    atomicAdd(&C[row * C_COLS + 3], a_element * thread_elements[3]);
    atomicAdd(&C[row * C_COLS + 4], a_element * thread_elements[4]);
    atomicAdd(&C[row * C_COLS + 5], a_element * thread_elements[5]);
    atomicAdd(&C[row * C_COLS + 6], a_element * thread_elements[6]);
    atomicAdd(&C[row * C_COLS + 7], a_element * thread_elements[7]);
  }
}

void matrixMultiplyCPU(float* A, float* B, float* C) {
  for (int row = 0; row < A_ROWS; row++) {
    for (int col = 0; col < B_COLS; col++) {
      float sum = 0.0f;
      for (int i = 0; i < A_COLS; i++) {
        sum += A[row * A_COLS + i] * B[i * B_COLS + col];
      }
      C[row * C_COLS + col] = sum;
    }
  }
}

// Verify results
bool verifyResults(float* gpuResult, float* cpuResult, int size, float tolerance = 1e-5) {
  for (int i = 0; i < size; i++) {
    if (fabs(gpuResult[i] - cpuResult[i]) > tolerance) {
      printf("Mismatch at position %d: GPU = %f, CPU = %f\n", 
          i, gpuResult[i], cpuResult[i]);
      return false;
    }
  }
  return true;
}

int main() {
  float *h_A = (float*)malloc(A_ROWS * A_COLS * sizeof(float));
  float *h_B = (float*)malloc(B_ROWS * B_COLS * sizeof(float));
  float *h_C = (float*)malloc(C_ROWS * C_COLS * sizeof(float));
  float *h_C_cpu = (float*)malloc(C_ROWS * C_COLS * sizeof(float));

  randomize_matrix(h_A, A_ROWS, A_COLS);
  randomize_matrix(h_B, B_ROWS, B_COLS);

  float *d_A, *d_B, *d_C;
  cudaCheckError(hipMalloc(&d_A, A_ROWS * A_COLS * sizeof(float)));
  cudaCheckError(hipMalloc(&d_B, B_ROWS * B_COLS * sizeof(float)));
  cudaCheckError(hipMalloc(&d_C, C_ROWS * C_COLS * sizeof(float)));

  cudaCheckError(hipMemcpy(d_A, h_A, A_ROWS * A_COLS * sizeof(float), hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(d_B, h_B, B_ROWS * B_COLS * sizeof(float), hipMemcpyHostToDevice));

  hipFree(0);
  hipMemset(d_C, 0, C_ROWS * C_COLS * sizeof(float));

  dim3 gridDim(32,16,2);
  dim3 blockDim(32,1);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  matrixMultiplyKernel<<<gridDim, blockDim>>>(d_A, d_B, d_C,4,512);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  cudaCheckError(hipMemcpy(h_C, d_C, C_ROWS * C_COLS * sizeof(float), hipMemcpyDeviceToHost));

  float time = 0.0f;
  hipEventElapsedTime(&time, start, stop);

  std::cout << "GPU Timing: " << time << " ms" << std::endl;
  matrixMultiplyCPU(h_A, h_B, h_C_cpu);

  bool correct = verifyResults(h_C, h_C_cpu, C_ROWS * C_COLS);

  free(h_A);
  free(h_B);
  free(h_C);
  free(h_C_cpu);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return 0;
}
