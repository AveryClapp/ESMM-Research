#include "hip/hip_runtime.h"
#pragma once

/* Preprocessor for A matrix to encode horizontal sparsity */

#include "utils.cuh"
#include <hip/hip_runtime.h>

template <const int BM, const int BN, const int BK, const int WM, const int WN,
		const int WNITER, const int TM, const int TN, const int NUM_THREADS>
__global__ void __launch_bounds__(NUM_THREADS)
	preprocess_A(int M, int N, int K, float *A, int *A_CTS, int* A_LIST) {
	const uint cRow = blockIdx.y;
	const uint cCol = blockIdx.x;

	const uint warpIdx = threadIdx.x / WARPSIZE;
	const uint warpCol = warpIdx % (BN / WN);
	const uint warpRow = warpIdx / (BN / WN);

	constexpr uint WMITER = (WM * WN) / (WARPSIZE * TM * TN * WNITER);
	constexpr uint WSUBM = WM / WMITER;

	const uint threadIdxInWarp = threadIdx.x % WARPSIZE;
	const uint threadColInWarp = threadIdxInWarp % (WSUBN / TN); 
	const uint threadRowInWarp = threadIdxInWarp / (WSUBN / TN); 

	__shared__ float As[BN * BK];

	A += cRow * BM * K;

	const uint innerRowA = threadIdx.x / (BK / 4);
	const uint innerColA = threadIdx.x % (BK / 4);
	constexpr uint rowStrideA = (NUM_THREADS * 4) / BK;

	for (int32_t bkIdx = 0; bkIdx < K; bkIdx += BK) {
		for (int32_t offset = 0; offset + rowStrideA <= BM; offset += rowStrideA) {
			const float4 tmp = reinterpret_cast<const float4 *>(
				&A[(innerRowA + offset) * K + innerColA * 4])[0];
			As[(innerColA * 4 + 0) * BM + innerRowA + offset] = tmp.x;
			As[(innerColA * 4 + 1) * BM + innerRowA + offset] = tmp.y;
			As[(innerColA * 4 + 2) * BM + innerRowA + offset] = tmp.z;
			As[(innerColA * 4 + 3) * BM + innerRowA + offset] = tmp.w;
		}

		__syncthreads();
		__shared__ int A_cols[BK * WMITER];
		for (int8_t dotIdx = 0; dotIdx < BK; ++dotIdx) {
			short cts = 0;
			for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
				regM[wSubRowIdx] = As[(dotIdx * BM) + warpRow * WM +
					wSubRowIdx * WSUBM + threadRowInWarp];
			}

			for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
				uint32_t ballot = __ballot_sync(0xFFFFFFFF, regM[wSubRowIdx]);
				int isDense = (ballot != 0);
				denseList[idx] = (dotIdx << 2) | wSubRowIdx;
				denseCount += isDense & -(laneId == 0);
				idx += isDense;
			}
		}
		A += BK;
		__syncthreads();
	}

	for (uint wSubRowIdx = 0; wSubRowIdx < WMITER; ++wSubRowIdx) {
		for (uint wSubColIdx = 0; wSubColIdx < WNITER; ++wSubColIdx) {
			float *C_interim = C + (wSubRowIdx * WSUBM) * N + wSubColIdx * WSUBN;
			for (uint resIdxM = 0; resIdxM < TM; resIdxM += 1) {
				for (uint resIdxN = 0; resIdxN < TN; resIdxN += 4) {
					float4 tmp;
					const int i = (wSubRowIdx * TM + resIdxM) * (WNITER * TN) +
						wSubColIdx * TN + resIdxN;
					tmp.x = threadResults[i + 0];
					tmp.y = threadResults[i + 1];
					tmp.z = threadResults[i + 2];
					tmp.w = threadResults[i + 3];
					reinterpret_cast<float4 *>(
						&C_interim[(threadRowInWarp * TM + resIdxM) * N +
						threadColInWarp * TN + resIdxN])[0] = tmp;
				}
			}
		}	
	}		
}

